#include "hip/hip_runtime.h"
#include "System.h"
#include "SystemStructures.h" 
#include "AreaTriangles.h"
#include "AreaTrianglesEnergy.h"
#include "BendingTriangles.h"
#include "BendingTrianglesEnergy.h"
#include "MemRepulsionSprings.h"
#include "MemRepulsionEnergy.h"
#include "LinearSprings.h"
#include "LinearSpringsEnergy.h"
#include "LJSprings.h"
#include "LJSprings_LJ.h"
//#include "LJEnergy.h"
#include "NodeAdvance.h"
#include "BucketScheme.h"
#include "Storage.h" 
#include "Edgeswap_test.h"
#include "SystemBuilder.h"
#include <vector>
#include "VolumeComp.h"
#include "VolumeSprings.h"
#include <bits/stdc++.h>
#include "LineTensionSprings.h"

 //somehow the gradient is not being set in my version

//bool IsPos (int i){return (i>=0);}
int count_bigger(const std::vector<int>& elems) {
    return std::count_if(elems.begin(), elems.end(), [](int c){return c >= 0;});
}

System::System() {};

void System::Solve_Forces(){

	thrust::fill(coordInfoVecs.nodeForceX.begin(), coordInfoVecs.nodeForceX.end(), 0.0);
	thrust::fill(coordInfoVecs.nodeForceY.begin(), coordInfoVecs.nodeForceY.end(), 0.0);
	thrust::fill(coordInfoVecs.nodeForceZ.begin(), coordInfoVecs.nodeForceZ.end(), 0.0);
	
	//setBucketScheme();
	//std::cout<<"Error here! before linear"<<std::endl;
	ComputeLinearSprings( 
		generalParams, 
		coordInfoVecs,
		linearSpringInfoVecs, 
		ljInfoVecs);
	//	std::cout<<"LINEAR"<<std::endl;
	
	//if (coordInfoVecs.nodeLocX.size() > 162){
	//	std::cout<<"force ="<<coordInfoVecs.nodeForceX[generalParams.maxNodeCount-1]<<" "<<coordInfoVecs.nodeForceY[generalParams.maxNodeCount-1]<<" "<<coordInfoVecs.nodeForceZ[generalParams.maxNodeCount-1]<<std::endl;
	//}
	//std::cout<<"Error here! before area"<<std::endl;
	ComputeAreaTriangleSprings(
		generalParams,
		coordInfoVecs,
		areaTriangleInfoVecs);
	//	std::cout<<"AREA"<<std::endl;
	
	//std::cout<<"Error here! before bending"<<std::endl;
	ComputeCosTriangleSprings(
		generalParams,
		coordInfoVecs,  
		bendingTriangleInfoVecs); 
	
	//std::cout<<"Error here! before memrepul"<<std::endl;
	ComputeMemRepulsionSprings(
		coordInfoVecs,
		linearSpringInfoVecs, 
		capsidInfoVecs,
		generalParams,
		auxVecs);

	ComputeVolume(
		generalParams,
		coordInfoVecs,
		linearSpringInfoVecs,
		ljInfoVecs);

	ComputeVolumeSprings(
		coordInfoVecs,
		linearSpringInfoVecs, 
		capsidInfoVecs,
		generalParams,
		auxVecs);

	ComputeLineTensionSprings(
		generalParams,
		coordInfoVecs,
		linearSpringInfoVecs);
		
};


void System::solveSystem() {
	double pull_strength = 4.0;
	int translate_frequency = 10;
	double beta1 = 0.0;
	double beta2 = 0.0;
	double EXPAN_THRESHOLD = 2.0;
	std::cout<<"EXPANSION THRESHOLD = "<<EXPAN_THRESHOLD<<std::endl;

	double displacementX, displacementY, displacementZ;
	double newcenterX, newcenterY, newcenterZ;

	//coordInfoVecs.num_triangles = coordInfoVecs.triangles2Nodes_1.size();
	//generalParams.maxNodeCount = coordInfoVecs.nodeLocX.size();
	//generalParams.num_of_edges = coordInfoVecs.edges2Nodes_1.size();
	//generalParams.eq_total_volume = 200.0;
	std::vector<int> VectorShuffleForGrowthLoop;
	////IDENTIFY CENTER OF THE SPHERE////////////////////////////////////////
	////this is necessary to choose where to generate new lj points//////////
	////which will be located within a distance away from the center/////////
	for (int i = 0; i < generalParams.maxNodeCount; i++){
		generalParams.centerX += coordInfoVecs.nodeLocX[i];
		generalParams.centerY += coordInfoVecs.nodeLocY[i];
		generalParams.centerZ += coordInfoVecs.nodeLocZ[i];
	}
	generalParams.centerX = generalParams.centerX/generalParams.maxNodeCount;
	generalParams.centerY = generalParams.centerY/generalParams.maxNodeCount;
	generalParams.centerZ = generalParams.centerZ/generalParams.maxNodeCount;
	////////////////////////////////////////////////////////////////////////
	////////////////////////////////////////////////////////////////////////

	std::vector<double> V1 = {0.0, 0.0, 0.7071, -0.7071, 0.0, 0.0};/*{ 0.2294,   -0.7103,    1.21788 ,  -1.3525 ,  -0.3047,
		0.8073,   -0.9284 ,  -0.9918 ,  -1.7507 ,  -2.1332,
		1.1312 ,  -0.2897 ,  -1.0996 ,  -0.3552 ,   0.9047};*/ //{-0.02283, -0.02283, -0.02283};

	std::vector<double> V2 = {0.0, 0.0, 0.0, 0.0, 0.7071, -0.7071};/*{-0.7403 ,   0.0554 ,  -0.21368  ,  1.0354 ,   2.0994,
		0.1924 ,  -1.7749 ,   0.4700 ,   0.5478 ,  -1.3549,
		0.4430 ,   0.6601 ,  -0.6469 ,  -1.0153 ,   0.9085};*/ //{2.384208, 2.384208, 1.68};

	std::vector<double> V3 = {0.7071, -0.7071, 0.0, 0.0, 0.0, 0.0};/*{-0.9997 ,  -0.8749 ,  -1.82367 ,  -0.7812,    0.4490,
		2.0808 ,  -0.2730 ,   0.4642 ,   1.1056  , 0.6132,
		0.3975 ,  -0.9000 ,   2.1327  , -0.8614 ,  -0.6783};*/ //{1.5243396, 1.1043396, 1.5243396};
	
	
	for (int i = 0; i < V1.size(); i++){
		ljInfoVecs.LJ_PosX_all.push_back(V1[i]); 
		ljInfoVecs.LJ_PosY_all.push_back(V2[i]);
		ljInfoVecs.LJ_PosZ_all.push_back(V3[i]);
	}  
	//ljInfoVecs.LJ_PosX_all.push_back();
	//ljInfoVecs.LJ_PosY_all.push_back();
	//ljInfoVecs.LJ_PosZ_all.push_back();

	ljInfoVecs.forceX_all.reserve(ljInfoVecs.LJ_PosX_all.size());
	ljInfoVecs.forceY_all.reserve(ljInfoVecs.LJ_PosX_all.size());
	ljInfoVecs.forceZ_all.reserve(ljInfoVecs.LJ_PosX_all.size());

	generalParams.maxNodeCountLJ = ljInfoVecs.LJ_PosX_all.size();


	std::vector<int> out;
	//int ALPHA;

	std::vector<bool> boundary_edges;
	boundary_edges.reserve(coordInfoVecs.num_edges);
	for (int i = 0; i < coordInfoVecs.num_edges; i++){
		if (coordInfoVecs.edges2Triangles_1[i] == coordInfoVecs.edges2Triangles_2[i]){
			boundary_edges.push_back(true);
		}
		else {
			boundary_edges.push_back(false);
		}
	}

	std::vector<int> edgeIndices;
	edgeIndices.reserve(coordInfoVecs.num_edges);
	for (int i = 0; i < coordInfoVecs.num_edges; ++i){
		//edgeIndices.push_back(edge_to_ljparticle[i]);
		if (boundary_edges[i] == false){
			edgeIndices.push_back(i);
		}
		else {
			edgeIndices.push_back(-1);
		}
	}



	auto it = remove_if(edgeIndices.begin(), edgeIndices.end(),  [](const int i) {return i < 0; });
	edgeIndices.erase(it, edgeIndices.end());

	//std::vector<int> nodes_to_center;
	generalParams.nodes_in_upperhem.resize(generalParams.maxNodeCount);
	for (int i = 0; i < generalParams.maxNodeCount; i++){
		if (coordInfoVecs.nodeLocZ[i] > (generalParams.centerZ + 5.5)){
			generalParams.nodes_in_upperhem[i] = 1;
		}
		else{
			generalParams.nodes_in_upperhem[i] = -1;
		}
	//	std::cout<<"nodes "<<i<<" "<<generalParams.nodes_in_upperhem[i]<<std::endl;		
	}

	////////////////////////////////////////////////////////////////////////////////////////////



	////////////////////////////////////////////////////////////////////////////////////////////
	generalParams.triangles_in_upperhem.resize(coordInfoVecs.num_triangles);
	for (int i = 0; i < coordInfoVecs.num_triangles; i++){
		int aaa = generalParams.nodes_in_upperhem[coordInfoVecs.triangles2Nodes_1[i]];
		//std::cout<<aaa<<std::endl;
		int bbb = generalParams.nodes_in_upperhem[coordInfoVecs.triangles2Nodes_2[i]];
		//std::cout<<bbb<<std::endl;
		int ccc = generalParams.nodes_in_upperhem[coordInfoVecs.triangles2Nodes_3[i]];
		//std::cout<<ccc<<std::endl;
		if ((aaa+bbb+ccc)==3){
			generalParams.triangles_in_upperhem[i] = 1;
			//triangles_in_upperhem.push_back(i);
		}
		else if ((aaa+bbb+ccc)==1){
			generalParams.triangles_in_upperhem[i] = 0;
			//triangles_in_upperhem.push_back(i);
		}
		else{
			generalParams.triangles_in_upperhem[i] = -1;
		}
	//	std::cout<<"triangle "<<i<<" "<<generalParams.triangles_in_upperhem[i]<<std::endl;		
	}

	std::vector<int> edges_in_upperhem;
	generalParams.edges_in_upperhem.resize(coordInfoVecs.num_edges);
	for (int i = 0; i < coordInfoVecs.num_edges; i++){
		int aaa = generalParams.triangles_in_upperhem[coordInfoVecs.edges2Triangles_1[i]];//generalParams.nodes_in_upperhem[coordInfoVecs.edges2Nodes_1[i]];
		int bbb = generalParams.triangles_in_upperhem[coordInfoVecs.edges2Triangles_2[i]];//generalParams.nodes_in_upperhem[coordInfoVecs.edges2Nodes_2[i]];
		if (aaa == 1 && bbb == 1){
			generalParams.edges_in_upperhem[i] = 1;
			edges_in_upperhem.push_back(i);
		}
		else if (aaa == 1 || bbb == 1){
			generalParams.edges_in_upperhem[i] = 0;
		}
		else{
			generalParams.edges_in_upperhem[i] = -1;
		}
		
	}
	

	//Find the boundary of the nodes_in_upperhem region
	generalParams.boundaries_in_upperhem.resize(coordInfoVecs.num_edges);
	for (int i = 0; i < coordInfoVecs.num_edges; i++){
		double T1 = coordInfoVecs.edges2Triangles_1[i];
		double T2 = coordInfoVecs.edges2Triangles_2[i];
		if (generalParams.triangles_in_upperhem[T1] == 1 && generalParams.triangles_in_upperhem[T2] == 0){
			generalParams.boundaries_in_upperhem[i] = 1;
		}
		else if (generalParams.triangles_in_upperhem[T1] == 0 && generalParams.triangles_in_upperhem[T2] == 1){
			generalParams.boundaries_in_upperhem[i] = 1;
		}
		else {
			generalParams.boundaries_in_upperhem[i] = -1;
		}
	}
	generalParams.eq_total_boundary_length = generalParams.boundaries_in_upperhem.size()*generalParams.Rmin;
	

	//std::vector<int> edge_to_ljparticle;
	//generalParams.edge_to_ljparticle.reserve(coordInfoVecs.num_edges);
	for (int i = 0; i < coordInfoVecs.num_edges; i++){
		generalParams.edge_to_ljparticle.push_back(-1);
	};
	
	//int node1, node2;
	//double R1, R2;
	//int last_index;
	//double Influence_Range = ljInfoVecs.Rmin - 0.5;
	//std::cout<<"Influence_Range = "<<Influence_Range<<std::endl;
	int num_edge_loop;
	//double LJ_PosX_backup, LJ_PosY_backup, LJ_PosZ_backup;
	
	double Max_Runtime = 0.0020;
	double Max_RunStep = Max_Runtime/generalParams.dt;
	std::cout<<"Max runtime = "<<Max_Runtime<<std::endl;
	std::cout<<"Max runstep = "<<Max_RunStep<<std::endl;
	//linearSpringInfoVecs.spring_constant_att1 = 50.0;
	//linearSpringInfoVecs.spring_constant_att2 = 0.75;
	linearSpringInfoVecs.spring_constant_rep1 = 0.01;
	linearSpringInfoVecs.spring_constant_rep2 = 9.0;
	//std::cout<<"spring_constnat_att1 = "<<linearSpringInfoVecs.spring_constant_att1<<std::endl;
	//std::cout<<"spring_constnat_att2 = "<<linearSpringInfoVecs.spring_constant_att2<<std::endl;
	//std::cout<<"spring_constnat_rep1 = "<<linearSpringInfoVecs.spring_constant_rep1<<std::endl;
	//std::cout<<"spring_constnat_rep2 = "<<linearSpringInfoVecs.spring_constant_rep2<<std::endl;

	generalParams.volume_spring_constant = 1.0;//1.25;
	std::cout<<"volume spring constant = "<<generalParams.volume_spring_constant<<std::endl;
	generalParams.line_tension_constant = 40.0;
	std::cout<<"line tension constant = "<<generalParams.line_tension_constant<<std::endl;

	double scale_linear = 30.0;
	double scale_bend = 30.0;
	double scale_area = 30.0;
	std::cout<<"scaling of different region linear = "<<scale_linear<<std::endl;
	std::cout<<"scaling of different region bend = "<<scale_bend<<std::endl;
	std::cout<<"scaling of different region area = "<<scale_area<<std::endl;
	linearSpringInfoVecs.spring_constant_weak = linearSpringInfoVecs.spring_constant/scale_linear;
	bendingTriangleInfoVecs.spring_constant_weak = bendingTriangleInfoVecs.spring_constant/scale_bend;
	areaTriangleInfoVecs.spring_constant_weak = areaTriangleInfoVecs.spring_constant/scale_area;
	
	//areaTriangleInfoVecs.initial_area = 0.6;

	bendingTriangleInfoVecs.initial_angle = 0.0906;//0.17549;//0.15;//0.0906;
	bendingTriangleInfoVecs.initial_angle_raft = 0.0906;//0.17549;//0.15;
	bendingTriangleInfoVecs.initial_angle_coat = 0.0906;//0.17549;//0.15;//0.167448079;
	
	bendingTriangleInfoVecs.spring_constant_raft = 0.0;//bendingTriangleInfoVecs.spring_constant;
	bendingTriangleInfoVecs.spring_constant_coat = 0.0;//bendingTriangleInfoVecs.spring_constant;
	bendingTriangleInfoVecs.spring_constant = bendingTriangleInfoVecs.spring_constant*(2/sqrt(3));
	bendingTriangleInfoVecs.spring_constant_raft = bendingTriangleInfoVecs.spring_constant_raft*(2/sqrt(3));
	bendingTriangleInfoVecs.spring_constant_coat = bendingTriangleInfoVecs.spring_constant_coat*(2/sqrt(3));
	std::cout<<"effective bending = "<<bendingTriangleInfoVecs.spring_constant<<std::endl;
	std::cout<<"effective bending raft = "<<bendingTriangleInfoVecs.spring_constant_raft<<std::endl;
	std::cout<<"effective bending coat = "<<bendingTriangleInfoVecs.spring_constant_coat<<std::endl;
	
	//std::cout<<"coat angle = "<<bendingTriangleInfoVecs.initial_angle_coat<<std::endl;
	generalParams.Rmin = 1.0;
	generalParams.abs_Rmin = 0.75;//0.586955;
	ljInfoVecs.Rmin_M = 2.0;
	ljInfoVecs.Rcutoff_M = 5.9;
	ljInfoVecs.Rmin_LJ = 1.0;//3.0//1.0;
	ljInfoVecs.Rcutoff_LJ = 1.25;//3.0;//1.0;
	//ljInfoVecs.epsilon_M = 1.0;
	ljInfoVecs.epsilon_M_att1 = 0.0;//6.0;//16.0;
	ljInfoVecs.epsilon_M_att2 = 0.0;//1.0;//1.0;
	std::cout<<"Morse_NM_D_att = "<<ljInfoVecs.epsilon_M_att1<<std::endl;
	std::cout<<"Morse_NM_a_att = "<<ljInfoVecs.epsilon_M_att2<<std::endl;
	ljInfoVecs.epsilon_M_rep1 = 12.5;//16.0;
	ljInfoVecs.epsilon_M_rep2 = 0.75;//1.0;
	std::cout<<"Morse_NM_D_rep = "<<ljInfoVecs.epsilon_M_rep1<<std::endl;
	std::cout<<"Morse_NM_a_rep = "<<ljInfoVecs.epsilon_M_rep2<<std::endl;
	//ljInfoVecs.epsilon_LJ = 0.25;
	ljInfoVecs.epsilon_LJ_rep1 = 10.0;//0.5;// 0.06;//7.5;
	ljInfoVecs.epsilon_LJ_rep2 = 0.75;//1.0;//1.0;//1.0;
	std::cout<<"Morse_NN_D = "<<ljInfoVecs.epsilon_LJ_rep1<<std::endl;
	std::cout<<"Morse_NN_a = "<<ljInfoVecs.epsilon_LJ_rep2<<std::endl;
	//std::cout<<"Absolute minimum edge size = "<<generalParams.abs_Rmin<<std::endl;
	//std::cout<<"Morse D = "<<linearSpringInfoVecs.spring_constant_rep1<<std::endl;
	//std::cout<<"Morse a = "<<linearSpringInfoVecs.spring_constant_rep2<<std::endl;
	std::cout<<"spontaneous angle = "<<bendingTriangleInfoVecs.initial_angle<<std::endl;

	double initial_kT;
	initial_kT = generalParams.kT;//This is for the acceptance of change after looping through every edge within proximity.
	//	double SAMPLE_SIZE = 0.02;
	//std::cout<<"Sample size: "<<SAMPLE_SIZE<<std::endl;
	auto edgeswap_ptr = std::make_shared<Edgeswap>(coordInfoVecs, generalParams);


	bool runSim = true;

	int GROWTH_TIME = 1;
	int RECORD_TIME = 50;//round(Max_RunStep/2);
	std::cout<<"Record frequency = "<<RECORD_TIME<<std::endl;
	std::cout<<"Growth frequency = "<<GROWTH_TIME<<std::endl;
	int NKBT = 10000; //The max number of edge-swap attempt per kBT value
	std::cout<<"Number of edge-swap per kBT value = "<<NKBT<<std::endl;
	double min_kT = 0.21;
	std::cout<<"min kT for sim. termination = "<<min_kT<<std::endl;
	int WHEN = 0;
	double old_total_energy = 0.0;
	double new_total_energy = 0.0;
	double energy_gradient = 0.0;
	int Num_of_step_run = 0;
	auto build_ptr = weak_bld_ptr.lock();//upgrade weak builder to access host variables.
	std::cout<<"initial LJ-x : "<< ljInfoVecs.LJ_PosX <<std::endl;
	std::cout<<"initial LJ-y : "<< ljInfoVecs.LJ_PosY <<std::endl;
	std::cout<<"initial LJ-z : "<< ljInfoVecs.LJ_PosZ <<std::endl;
		

    
	double min_energy;
	generalParams.true_num_edges = 0;
	for (int i = 0; i < coordInfoVecs.num_edges; i++){
		if (coordInfoVecs.edges2Nodes_1[i] != INT_MAX && coordInfoVecs.edges2Nodes_2[i] != INT_MAX){
			generalParams.true_num_edges += 1;
		}
	}
	//storage->print_VTK_File();
	////storage->storeVariables();

	////////////////////////////////////////
	
	ComputeVolume(
		generalParams,
		coordInfoVecs,
		linearSpringInfoVecs,
		ljInfoVecs
	);

	generalParams.eq_total_volume = generalParams.true_current_total_volume*1.5;//This is for setting different equilibrium volume to mimic growth or shirnkage.
	std::cout<<"true_current_total_volume = "<<generalParams.true_current_total_volume<<std::endl;
	std::cout<<"eq_total_volume = "<<generalParams.eq_total_volume<<std::endl;

	////////////////////////////////////////
	
	while (runSim == true){
		//WHEN += 1;
		double current_time = 0.0;

		//generalParams.kT = 1.0;//reset kT before simulations starts.
		//Max_Runtime = 0.0;//2.5;
		int translate_counter = 0;
			while (current_time < (Max_Runtime)){
					translate_counter += 1;
					Solve_Forces();
				
					double energy_rep =
					ComputeMemRepulsionEnergy(
						coordInfoVecs,
						linearSpringInfoVecs, 
						capsidInfoVecs,
						generalParams,
						auxVecs);

					//now forces are computed, move nodes.
					
					

					for (int i = 0; i < ljInfoVecs.LJ_PosX_all.size(); i++){

						ljInfoVecs.LJ_PosX = ljInfoVecs.LJ_PosX_all[i];
				//		std::cout<<"LJ_PosX = "<<ljInfoVecs.LJ_PosX<<std::endl;
						ljInfoVecs.LJ_PosY = ljInfoVecs.LJ_PosY_all[i];
				//		std::cout<<"LJ_PosY = "<<ljInfoVecs.LJ_PosY<<std::endl;
						ljInfoVecs.LJ_PosZ = ljInfoVecs.LJ_PosZ_all[i];
						
						
					
							ComputeLJSprings(
								coordInfoVecs,
								ljInfoVecs,
								generalParams);
							ljInfoVecs.forceX_all[i] =  ljInfoVecs.forceX;
							ljInfoVecs.forceY_all[i] =  ljInfoVecs.forceY;
							ljInfoVecs.forceZ_all[i] =  ljInfoVecs.forceZ;						

						ComputeLJSprings_LJ(
							coordInfoVecs,
							ljInfoVecs,
							generalParams);
						ljInfoVecs.forceX_all[i] +=  ljInfoVecs.forceX;
						ljInfoVecs.forceY_all[i] +=  ljInfoVecs.forceY;
						ljInfoVecs.forceZ_all[i] +=  ljInfoVecs.forceZ;	
						if (i == 0){
							double R = sqrt( 
								(ljInfoVecs.LJ_PosX - coordInfoVecs.nodeLocX[35]) * (ljInfoVecs.LJ_PosX - coordInfoVecs.nodeLocX[35]) + 
								(ljInfoVecs.LJ_PosY - coordInfoVecs.nodeLocY[35]) * (ljInfoVecs.LJ_PosY - coordInfoVecs.nodeLocY[35]) + 
								(ljInfoVecs.LJ_PosZ - coordInfoVecs.nodeLocZ[35]) * (ljInfoVecs.LJ_PosZ - coordInfoVecs.nodeLocZ[35]) );
							double magnitude = -2.0*(pull_strength/2.0)*(R - ljInfoVecs.Rmin_M)*(1.0/R);
												//2*40.0*(1-exp(-1.5*(R-ljInfoVecs.Rmin_M)))*
												//(-exp(-1.5*(R-ljInfoVecs.Rmin_M)))*
												//(1.0/R);
							double forceX = -magnitude*(ljInfoVecs.LJ_PosX - coordInfoVecs.nodeLocX[35]);//xLoc_LR;
							double forceY = -magnitude*(ljInfoVecs.LJ_PosY - coordInfoVecs.nodeLocY[35]);//yLoc_LR;
							double forceZ = -magnitude*(ljInfoVecs.LJ_PosZ - coordInfoVecs.nodeLocZ[35]);//zLoc_LR;
							ljInfoVecs.forceX_all[i] +=  -forceX;
							ljInfoVecs.forceY_all[i] +=  -forceY;
							ljInfoVecs.forceZ_all[i] +=  -forceZ;
							//coordInfoVecs.nodeForceX[35] += forceX;
							//coordInfoVecs.nodeForceY[35] += forceY;
							//coordInfoVecs.nodeForceZ[35] += forceZ;

							}				
						
					}

					double beta;
					for (int i = 0; i < ljInfoVecs.LJ_PosX_all.size(); i++){
						
						if(i == 0){
							beta = beta1;
						}
						else{
							beta = beta2;
						}
						ljInfoVecs.LJ_PosX_all[i] = ljInfoVecs.LJ_PosX_all[i] + generalParams.dt * ljInfoVecs.forceX_all[i];
						ljInfoVecs.LJ_PosY_all[i] = ljInfoVecs.LJ_PosY_all[i] + generalParams.dt * ljInfoVecs.forceY_all[i];
						ljInfoVecs.LJ_PosZ_all[i] = ljInfoVecs.LJ_PosZ_all[i] + generalParams.dt * (ljInfoVecs.forceZ_all[i] + beta);
					
					}

				AdvancePositions(
					coordInfoVecs,
					generalParams,
					domainParams);
				if (translate_counter % translate_frequency == 1){

					newcenterX = 0.0;
					newcenterY = 0.0;
					newcenterZ = 0.0;
					for (int i = 0; i < coordInfoVecs.nodeLocX.size(); i++){
						newcenterX += coordInfoVecs.nodeLocX[i];
						newcenterY += coordInfoVecs.nodeLocY[i];
						newcenterZ += coordInfoVecs.nodeLocZ[i];
					}
					newcenterX = newcenterX/coordInfoVecs.nodeLocX.size();
					newcenterY = newcenterY/coordInfoVecs.nodeLocX.size();
					newcenterZ = newcenterZ/coordInfoVecs.nodeLocX.size();
					displacementX = newcenterX - generalParams.centerX;
					displacementY = newcenterY - generalParams.centerY;
					displacementZ = newcenterZ - generalParams.centerZ;

					for (int i = 0; i < generalParams.maxNodeCount; i++){
						coordInfoVecs.nodeLocX[i] += -displacementX;
						coordInfoVecs.nodeLocY[i] += -displacementY;
						coordInfoVecs.nodeLocZ[i] += -displacementZ;
					}
					for (int i = 0; i < ljInfoVecs.LJ_PosX_all.size(); i++){
						ljInfoVecs.LJ_PosX_all[i] += -displacementX;
						ljInfoVecs.LJ_PosY_all[i] += -displacementY;
						ljInfoVecs.LJ_PosZ_all[i] += -displacementZ;
					}
				}
							
					new_total_energy = linearSpringInfoVecs.linear_spring_energy + 
						areaTriangleInfoVecs.area_triangle_energy + 
						bendingTriangleInfoVecs.bending_triangle_energy + 
						0.5*energy_rep + 
						ljInfoVecs.lj_energy_M +
						ljInfoVecs.lj_energy_LJ +
						generalParams.volume_energy;

				energy_gradient = sqrt((new_total_energy - old_total_energy)*(new_total_energy - old_total_energy));
				old_total_energy = new_total_energy;
				current_time+=generalParams.dt;
				

			}
		std::cout<<"current time (1st iter before edgeswap): "<< current_time << std::endl;
		std::cout<<"current total energy (1st iter before edgeswap) = "<<new_total_energy<<std::endl;
		std::cout<<"true_current_total_volume = "<<generalParams.true_current_total_volume<<std::endl;
		std::cout<<"eq_total_volume = "<<generalParams.eq_total_volume<<std::endl;
		std::cout<<"current KBT = "<<generalParams.kT<<std::endl;
		if (isnan(new_total_energy)==1){
			std::cout<<"Nan or Inf position update !!!!"<<std::endl;
			runSim = false;
			break;
		}
	
		//edgeswap_ptr->transferDtoH(coordInfoVecs, build_ptr->hostSetInfoVecs);//Currently this is treated as a backup of coordInfoVecs
		////storage->print_VTK_File();
		////storage->storeVariables();
		//runSim = false;
		//break;

		int edgeswap_iteration = 0;
		//double preswap_energy = new_total_energy;
		//double postswap_energy;
		//double Ediff = 0.0;
		//initial_kT = generalParams.kT;
		num_edge_loop = 10;//round(edges_in_upperhem.size()*SAMPLE_SIZE);	
		std::cout<<"num_edge_loop = "<<num_edge_loop<<std::endl;
	
 		while (initial_kT > 0){
 					////////////////////NOW RELAX THE ATTEMPTED EDGESWAP//////////////////////
					 current_time = 0.0;
					 translate_counter = 0;
 					while (current_time < Max_Runtime){
						 translate_counter += 1;
						 //std::cout<<"ERROR BEFORE RELAXATION"<<std::endl;
						 Solve_Forces();

 						double energy_rep =
 						ComputeMemRepulsionEnergy(
 							coordInfoVecs,
 							linearSpringInfoVecs, 
 							capsidInfoVecs,
 							generalParams,
							 auxVecs);
					
				
 						for (int i = 0; i < ljInfoVecs.LJ_PosX_all.size(); i++){
 							ljInfoVecs.LJ_PosX = ljInfoVecs.LJ_PosX_all[i];
 							ljInfoVecs.LJ_PosY = ljInfoVecs.LJ_PosY_all[i];
							 ljInfoVecs.LJ_PosZ = ljInfoVecs.LJ_PosZ_all[i];
							 
							
					
 							ComputeLJSprings(
 								coordInfoVecs,
 								ljInfoVecs,
 								generalParams);
 							ljInfoVecs.forceX_all[i] =  ljInfoVecs.forceX;
 							ljInfoVecs.forceY_all[i] =  ljInfoVecs.forceY;
							 ljInfoVecs.forceZ_all[i] =  ljInfoVecs.forceZ;

 							ComputeLJSprings_LJ(
 								coordInfoVecs,
 								ljInfoVecs,
 								generalParams);
 							ljInfoVecs.forceX_all[i] +=  ljInfoVecs.forceX;
 							ljInfoVecs.forceY_all[i] +=  ljInfoVecs.forceY;
							 ljInfoVecs.forceZ_all[i] +=  ljInfoVecs.forceZ;
							 
							 if (i == 0){
								double R = sqrt( 
									(ljInfoVecs.LJ_PosX - coordInfoVecs.nodeLocX[35]) * (ljInfoVecs.LJ_PosX - coordInfoVecs.nodeLocX[35]) + 
									(ljInfoVecs.LJ_PosY - coordInfoVecs.nodeLocY[35]) * (ljInfoVecs.LJ_PosY - coordInfoVecs.nodeLocY[35]) + 
									(ljInfoVecs.LJ_PosZ - coordInfoVecs.nodeLocZ[35]) * (ljInfoVecs.LJ_PosZ - coordInfoVecs.nodeLocZ[35]) );
								double magnitude = -2.0*(pull_strength/2.0)*(R - ljInfoVecs.Rmin_M)*(1.0/R);
													//2*40.0*(1-exp(-1.5*(R-ljInfoVecs.Rmin_M)))*
													//(-exp(-1.5*(R-ljInfoVecs.Rmin_M)))*
													//(1.0/R);
								double forceX = -magnitude*(ljInfoVecs.LJ_PosX - coordInfoVecs.nodeLocX[35]);//xLoc_LR;
								double forceY = -magnitude*(ljInfoVecs.LJ_PosY - coordInfoVecs.nodeLocY[35]);//yLoc_LR;
								double forceZ = -magnitude*(ljInfoVecs.LJ_PosZ - coordInfoVecs.nodeLocZ[35]);//zLoc_LR;
								ljInfoVecs.forceX_all[i] +=  -forceX;
								ljInfoVecs.forceY_all[i] +=  -forceY;
								ljInfoVecs.forceZ_all[i] +=  -forceZ;
								//coordInfoVecs.nodeForceX[35] += forceX;
								//coordInfoVecs.nodeForceY[35] += forceY;
								//coordInfoVecs.nodeForceZ[35] += forceZ;
	
								}	
							 
 						}
					
 						//now forces are computed, move nodes.
 						double beta;
						for (int i = 0; i < ljInfoVecs.LJ_PosX_all.size(); i++){
							
							if(i == 0){
								beta = beta1;
							}
							else{
								beta = beta2;
							}
							ljInfoVecs.LJ_PosX_all[i] = ljInfoVecs.LJ_PosX_all[i] + generalParams.dt * ljInfoVecs.forceX_all[i];
							ljInfoVecs.LJ_PosY_all[i] = ljInfoVecs.LJ_PosY_all[i] + generalParams.dt * ljInfoVecs.forceY_all[i];
							ljInfoVecs.LJ_PosZ_all[i] = ljInfoVecs.LJ_PosZ_all[i] + generalParams.dt * (ljInfoVecs.forceZ_all[i] + beta);
						
						}
						 
 						AdvancePositions(
 							coordInfoVecs,
 							generalParams,
							 domainParams);
						
						if (translate_counter % translate_frequency == 1){
							newcenterX = 0.0;
							newcenterY = 0.0;
							newcenterZ = 0.0;
							for (int i = 0; i < coordInfoVecs.nodeLocX.size(); i++){
								newcenterX += coordInfoVecs.nodeLocX[i];
								newcenterY += coordInfoVecs.nodeLocY[i];
								newcenterZ += coordInfoVecs.nodeLocZ[i];
							}
							newcenterX = newcenterX/coordInfoVecs.nodeLocX.size();
							newcenterY = newcenterY/coordInfoVecs.nodeLocX.size();
							newcenterZ = newcenterZ/coordInfoVecs.nodeLocX.size();
							displacementX = newcenterX - generalParams.centerX;
							displacementY = newcenterY - generalParams.centerY;
							displacementZ = newcenterZ - generalParams.centerZ;
			
							for (int i = 0; i < generalParams.maxNodeCount; i++){
							coordInfoVecs.nodeLocX[i] += -displacementX;
							coordInfoVecs.nodeLocY[i] += -displacementY;
							coordInfoVecs.nodeLocZ[i] += -displacementZ;
							}
							for (int i = 0; i < ljInfoVecs.LJ_PosX_all.size(); i++){
								ljInfoVecs.LJ_PosX_all[i] += -displacementX;
								ljInfoVecs.LJ_PosY_all[i] += -displacementY;
								ljInfoVecs.LJ_PosZ_all[i] += -displacementZ;
							}
						
						
							//std::cout<<"ERROR 1"<<std::endl;
							edgeswap_ptr->transferDtoH(coordInfoVecs, build_ptr->hostSetInfoVecs);
							//std::cout<<"ERROR 1.5"<<std::endl;
							for (int edge_loop = 0; edge_loop < num_edge_loop; edge_loop++) {
								//std::cout<<"edge_loop = "<<edge_loop<<std::endl;
								
								std::random_device rand_dev;
								std::mt19937 generator(rand_dev());
							   
							   std::uniform_int_distribution<int> distribution(1,edges_in_upperhem.size());
							   
							   int dice_roll = distribution(generator);
							   
							   int edge = edges_in_upperhem[dice_roll - 1];
							   
							   while (generalParams.boundaries_in_upperhem[edge] == 1 || edge == INT_MAX){
									dice_roll = distribution(generator);
									
									edge = edges_in_upperhem[dice_roll - 1];
							   }
							   //std::cout<<"edge = "<<edge<<std::endl;
								int ALPHA = edgeswap_ptr->edge_swap_host_vecs(
									edge,
									generalParams,
									build_ptr->hostSetInfoVecs,
									linearSpringInfoVecs,
									bendingTriangleInfoVecs,
									areaTriangleInfoVecs);
								
							}
							//NOTE: EDGESWAP ALGORITHM CURRENTLY IS WRITTEN TO ALLOW AT MOST 8 NEIGHBORING NODES PER NODE.
							//std::cout<<"ERROR 2"<<std::endl;
							edgeswap_ptr->transferHtoD(coordInfoVecs, build_ptr->hostSetInfoVecs);//Currently this is treated as a backup of coordInfoVecs
							//std::cout<<"ERROR 2.5"<<std::endl;
							
							
						}
						
 						new_total_energy = linearSpringInfoVecs.linear_spring_energy + 
 							areaTriangleInfoVecs.area_triangle_energy + 
 							bendingTriangleInfoVecs.bending_triangle_energy +
 							//0.5*energy_rep +
 							ljInfoVecs.lj_energy_M +  
							 ljInfoVecs.lj_energy_LJ +
							 generalParams.volume_energy;
 						//std::cout<<"new_total_energy = "<<new_total_energy<<std::endl;

 						energy_gradient = sqrt((new_total_energy - old_total_energy)*(new_total_energy - old_total_energy));
 						
 					old_total_energy = new_total_energy;
 					current_time+=generalParams.dt;
					 }				
											
			
 								
 					if (edgeswap_iteration % RECORD_TIME == 0){
						generalParams.true_num_edges = 0;
						for (int i = 0; i < coordInfoVecs.num_edges; i++){
							if (coordInfoVecs.edges2Nodes_1[i] != INT_MAX && coordInfoVecs.edges2Nodes_2[i] != INT_MAX){
								generalParams.true_num_edges += 1;
							}
						 }
 						storage->print_VTK_File();
						 std::cout<<"current total energy = "<< new_total_energy<<std::endl;
						 std::cout<<"true current total volume = "<<generalParams.true_current_total_volume<<std::endl;
 					}
 					if (edgeswap_iteration % NKBT == 0){
 						//storage->storeVariables();
					 }

					

					 edgeswap_iteration += 1;
					 
					/*if (edgeswap_iteration % GROWTH_TIME == 0){

						for (int i = 0; i < coordInfoVecs.nodeLocX.size(); i++){
							generalParams.centerX += coordInfoVecs.nodeLocX[i];
							generalParams.centerY += coordInfoVecs.nodeLocY[i];
							generalParams.centerZ += coordInfoVecs.nodeLocZ[i];
						}
						generalParams.centerX = generalParams.centerX/coordInfoVecs.nodeLocX.size();
						generalParams.centerY = generalParams.centerY/coordInfoVecs.nodeLocX.size();
						generalParams.centerZ = generalParams.centerZ/coordInfoVecs.nodeLocX.size();

						double x,y,z;
						std::random_device rand_dev0;
						std::mt19937 generator0(rand_dev0());
						std::uniform_real_distribution<double> guess(generalParams.centerX-1.0, generalParams.centerX+1.0);
						x = 5.0;//guess(generator0);
						y = 5.0;//guess(generator0);
						z = 5.0;//guess(generator0);
						bool goodchoice = false;
						double GAP;
						while (sqrt(x*x + y*y + z*z) > (2.0) && goodchoice == false){
							x = guess(generator0);
							y = guess(generator0);
							z = guess(generator0);
							if (sqrt(x*x + y*y + z*z) > 2.0){
								continue;
							}
							for (int i = 0; i < ljInfoVecs.LJ_PosX_all.size(); i++){
								GAP = sqrt((x-ljInfoVecs.LJ_PosX_all[i])*(x-ljInfoVecs.LJ_PosX_all[i]) +
											(y-ljInfoVecs.LJ_PosY_all[i])*(x-ljInfoVecs.LJ_PosY_all[i]) +
											(z-ljInfoVecs.LJ_PosZ_all[i])*(x-ljInfoVecs.LJ_PosZ_all[i]));
								if (GAP < 0.65){
									goodchoice = false;
									break;
								}
								else{goodchoice = true;}
							}
						}
						ljInfoVecs.LJ_PosX_all.push_back(x);
						ljInfoVecs.LJ_PosY_all.push_back(y);
						ljInfoVecs.LJ_PosZ_all.push_back(z);
						ljInfoVecs.forceX_all.resize(ljInfoVecs.LJ_PosX_all.size());
						ljInfoVecs.forceY_all.resize(ljInfoVecs.LJ_PosX_all.size());
						ljInfoVecs.forceZ_all.resize(ljInfoVecs.LJ_PosX_all.size());
						generalParams.maxNodeCountLJ = ljInfoVecs.LJ_PosX_all.size();
					}*/
 					//std::cout<<"edgeswap_iteration = "<<edgeswap_iteration<<std::endl;
 					if (edgeswap_iteration == NKBT){
 						generalParams.kT = -1.0;//generalParams.kT - 0.072;
 						std::cout<<"Current kBT = "<<generalParams.kT<<std::endl;
 						edgeswap_iteration = 0;
 					}
 					if (generalParams.kT < min_kT){
 						initial_kT = -1.0;
					runSim = false;
					break;
					 }

//std::cout<<"ERROR BEFORE GROWTH"<<std::endl;
/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
////////////////////////////////////// GROWTH OF THE CELL (MEMBRANE) ////////////////////////////////////////////////////
/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
// GROWTH ALGORITHM IS CURRENTLY WRITTEN TO ALLOW NO MORE THEN 8 NEIGHBORING NODES PER NODE /////////////////////////////

VectorShuffleForGrowthLoop.clear();
for (int i = 0; i < coordInfoVecs.num_edges; i++){
	VectorShuffleForGrowthLoop.push_back(i);
}
std::random_device rand_dev;
std::mt19937 generator2(rand_dev());
std::shuffle(std::begin(VectorShuffleForGrowthLoop), std::end(VectorShuffleForGrowthLoop), generator2);

//bool triggered = false;
int triggered_counter = 0;
for (int p = 0; p < VectorShuffleForGrowthLoop.size(); p++){
	//std::cout<<"p = "<<p<<std::endl;
		int k = VectorShuffleForGrowthLoop[p];
		if (coordInfoVecs. edges2Nodes_1[k] == INT_MAX || coordInfoVecs. edges2Nodes_2[k] == INT_MAX){
			continue;
		}
		//int k = p;
		//k -= triggered_counter;
		int iedge = k;
		//std::cout<<"node1 of iedge = "<<coordInfoVecs.edges2Nodes_1[iedge]<<std::endl;
		//std::cout<<"node2 of iedge = "<<coordInfoVecs.edges2Nodes_2[iedge]<<std::endl;
		int elem1 = coordInfoVecs.edges2Triangles_1[iedge];
		int elem2 = coordInfoVecs.edges2Triangles_2[iedge];
		//std::cout<<"elem1 of iedge = "<<elem1<<std::endl;
		//std::cout<<"elem2 of iedge = "<<elem2<<std::endl;
		////std::cout<<"GROWTH ERROR 1"<<std::endl;	
        int first_v = coordInfoVecs.triangles2Nodes_1[elem1];
        int second_v = coordInfoVecs.triangles2Nodes_2[elem1];
        int third_v = coordInfoVecs.triangles2Nodes_3[elem1];
        double v1x = coordInfoVecs.nodeLocX[second_v] - coordInfoVecs.nodeLocX[first_v];
        double v1y = coordInfoVecs.nodeLocY[second_v] - coordInfoVecs.nodeLocY[first_v];
        double v1z = coordInfoVecs.nodeLocZ[second_v] - coordInfoVecs.nodeLocZ[first_v];
        double v2x = coordInfoVecs.nodeLocX[third_v] - coordInfoVecs.nodeLocX[first_v];
        double v2y = coordInfoVecs.nodeLocY[third_v] - coordInfoVecs.nodeLocY[first_v];
        double v2z = coordInfoVecs.nodeLocZ[third_v] - coordInfoVecs.nodeLocZ[first_v];
        double This_area_v = sqrt((v1y*v2z - v2y*v1z)*(v1y*v2z - v2y*v1z) + 
                                ((-v1x*v2z) + (v2x*v1z))*((-v1x*v2z) + (v2x*v1z)) +
								(v1x*v2y - v2x*v1y)*(v1x*v2y - v2x*v1y))/2.0;
		int first_w = coordInfoVecs.triangles2Nodes_1[elem2];
        int second_w = coordInfoVecs.triangles2Nodes_2[elem2];
        int third_w = coordInfoVecs.triangles2Nodes_3[elem2];
        double w1x = coordInfoVecs.nodeLocX[second_w] - coordInfoVecs.nodeLocX[first_w];
        double w1y = coordInfoVecs.nodeLocY[second_w] - coordInfoVecs.nodeLocY[first_w];
        double w1z = coordInfoVecs.nodeLocZ[second_w] - coordInfoVecs.nodeLocZ[first_w];
        double w2x = coordInfoVecs.nodeLocX[third_w] - coordInfoVecs.nodeLocX[first_w];
        double w2y = coordInfoVecs.nodeLocY[third_w] - coordInfoVecs.nodeLocY[first_w];
        double w2z = coordInfoVecs.nodeLocZ[third_w] - coordInfoVecs.nodeLocZ[first_w];
        double This_area_w = sqrt((w1y*v2z - w2y*v1z)*(w1y*w2z - w2y*w1z) + 
                                ((-w1x*w2z) + (w2x*w1z))*((-w1x*w2z) + (w2x*w1z)) +
                                (w1x*w2y - w2x*w1y)*(w1x*w2y - w2x*w1y))/2.0;
        if ((This_area_v/ areaTriangleInfoVecs.initial_area >= EXPAN_THRESHOLD) || (This_area_w/ areaTriangleInfoVecs.initial_area >= EXPAN_THRESHOLD)){
            //triggered = true;
			triggered_counter += 1;
        }
        else{continue;}
		////std::cout<<"GROWTH ERROR 2"<<std::endl;	
		int t1e1, t1e2, t1e3, t2e1, t2e2, t2e3;

		if (coordInfoVecs.triangles2Edges_1[elem1] == iedge){
			t1e1 = coordInfoVecs.triangles2Edges_2[elem1];
			t1e2 = coordInfoVecs.triangles2Edges_3[elem1];
			//t1e3 = coordInfoVecs.triangles2Edges_1[elem1];
		}
		else if (coordInfoVecs.triangles2Edges_2[elem1] == iedge){
			t1e1 = coordInfoVecs.triangles2Edges_3[elem1];
			t1e2 = coordInfoVecs.triangles2Edges_1[elem1];
			//t1e3 = coordInfoVecs.triangles2Edges_2[elem1];
		} 
		else if (coordInfoVecs.triangles2Edges_3[elem1] == iedge){
			t1e1 = coordInfoVecs.triangles2Edges_1[elem1];
			t1e2 = coordInfoVecs.triangles2Edges_2[elem1];
			//t1e3 = coordInfoVecs.triangles2Edges_3[elem1];
		}
		////std::cout<<"GROWTH ERROR 3"<<std::endl;	

		if (coordInfoVecs.triangles2Edges_1[elem2] == iedge){
			t2e1 = coordInfoVecs.triangles2Edges_2[elem2];
			t2e2 = coordInfoVecs.triangles2Edges_3[elem2];
			//t2e3 = coordInfoVecs.triangles2Edges_1[elem2];
		}
		else if (coordInfoVecs.triangles2Edges_2[elem2] == iedge){
			t2e1 = coordInfoVecs.triangles2Edges_3[elem2];
			t2e2 = coordInfoVecs.triangles2Edges_1[elem2];
			//t2e3 = coordInfoVecs.triangles2Edges_2[elem2];
		} 
		else if (coordInfoVecs.triangles2Edges_3[elem2] == iedge){
			t2e1 = coordInfoVecs.triangles2Edges_1[elem2];
			t2e2 = coordInfoVecs.triangles2Edges_2[elem2];
			//t2e3 = coordInfoVecs.triangles2Edges_3[elem2];
		}
		//Note that in the above assignment, t1e3 and t2e3 are the edges shared by the neighboring triangles T1 and T2.
		////std::cout<<"GROWTH ERROR 4"<<std::endl;	

		
		int n1, n2, n3, n4;
		
		if ((coordInfoVecs.edges2Nodes_1[t1e1] == coordInfoVecs. edges2Nodes_1[iedge]) || (coordInfoVecs.edges2Nodes_1[t1e1] == coordInfoVecs. edges2Nodes_2[iedge]) ){
			n1 = coordInfoVecs.edges2Nodes_1[t1e1];
			n2 = coordInfoVecs.edges2Nodes_2[t1e1];
			if (coordInfoVecs.edges2Nodes_1[t1e1] == coordInfoVecs. edges2Nodes_1[iedge]){
				n3 = coordInfoVecs.edges2Nodes_2[iedge];
			}
			else if (coordInfoVecs.edges2Nodes_1[t1e1] == coordInfoVecs. edges2Nodes_2[iedge]){
				n3 = coordInfoVecs.edges2Nodes_1[iedge];
			}
		}
		else if ((coordInfoVecs.edges2Nodes_2[t1e1] == coordInfoVecs. edges2Nodes_1[iedge]) || (coordInfoVecs.edges2Nodes_2[t1e1] == coordInfoVecs. edges2Nodes_2[iedge]) ){
			n1 = coordInfoVecs.edges2Nodes_2[t1e1];
			n2 = coordInfoVecs.edges2Nodes_1[t1e1];
			if (coordInfoVecs.edges2Nodes_2[t1e1] == coordInfoVecs. edges2Nodes_1[iedge]){
				n3 = coordInfoVecs.edges2Nodes_2[iedge];
			}
			else if (coordInfoVecs.edges2Nodes_2[t1e1] == coordInfoVecs. edges2Nodes_2[iedge]){
				n3 = coordInfoVecs.edges2Nodes_1[iedge];
			}
		}
		////std::cout<<"GROWTH ERROR 5"<<std::endl;	

		if (coordInfoVecs.edges2Nodes_1[t2e1] == coordInfoVecs.edges2Nodes_1[iedge] || coordInfoVecs.edges2Nodes_1[t2e1] == coordInfoVecs.edges2Nodes_2[iedge]){
			n4 = coordInfoVecs.edges2Nodes_2[t2e1];
		}
		else if (coordInfoVecs.edges2Nodes_2[t2e1] == coordInfoVecs.edges2Nodes_1[iedge] || coordInfoVecs.edges2Nodes_2[t2e1] == coordInfoVecs.edges2Nodes_2[iedge]){
			n4 = coordInfoVecs.edges2Nodes_1[t2e1];
		}
		int safe_growth1 = 0;
		int safe_growth2 = 0;
		if (coordInfoVecs. nndata1[n2] >= 0){safe_growth1 += 1;        }
        if (coordInfoVecs. nndata2[n2] >= 0){safe_growth1 += 1;        }
        if (coordInfoVecs. nndata3[n2] >= 0){safe_growth1 += 1;        }
        if (coordInfoVecs. nndata4[n2] >= 0){safe_growth1 += 1;        }
        if (coordInfoVecs. nndata5[n2] >= 0){safe_growth1 += 1;        }
        if (coordInfoVecs. nndata6[n2] >= 0){safe_growth1 += 1;        }
        if (coordInfoVecs. nndata7[n2] >= 0){safe_growth1 += 1;        }
        if (coordInfoVecs. nndata8[n2] >= 0){safe_growth1 += 1;        }
        if (coordInfoVecs. nndata9[n2] >= 0){safe_growth1 += 1;        }
        if (coordInfoVecs. nndata10[n2] >= 0){safe_growth1 += 1;        }
        if (coordInfoVecs. nndata11[n2] >= 0){safe_growth1 += 1;        }
		if (coordInfoVecs. nndata12[n2] >= 0){safe_growth1 += 1;        }
		if (coordInfoVecs. nndata1[n4] >= 0){safe_growth2 += 1;        }
        if (coordInfoVecs. nndata2[n4] >= 0){safe_growth2 += 1;        }
        if (coordInfoVecs. nndata3[n4] >= 0){safe_growth2 += 1;        }
        if (coordInfoVecs. nndata4[n4] >= 0){safe_growth2 += 1;        }
        if (coordInfoVecs. nndata5[n4] >= 0){safe_growth2 += 1;        }
        if (coordInfoVecs. nndata6[n4] >= 0){safe_growth2 += 1;        }
        if (coordInfoVecs. nndata7[n4] >= 0){safe_growth2 += 1;        }
        if (coordInfoVecs. nndata8[n4] >= 0){safe_growth2 += 1;        }
        if (coordInfoVecs. nndata9[n4] >= 0){safe_growth2 += 1;        }
        if (coordInfoVecs. nndata10[n4] >= 0){safe_growth2 += 1;        }
        if (coordInfoVecs. nndata11[n4] >= 0){safe_growth2 += 1;        }
		if (coordInfoVecs. nndata12[n4] >= 0){safe_growth2 += 1;        }
		
		if (safe_growth1 >= 8 || safe_growth2 >= 8){
			continue;
		}

		//std::cout<<"n1 = "<<n1<<std::endl;
		//std::cout<<"n2 = "<<n2<<std::endl;
		//std::cout<<"n3 = "<<n3<<std::endl;
		//std::cout<<"n4 = "<<n4<<std::endl;
		//These extract the indices of vertices of the selected triangles "elem1" and "elem2". Now we have n1, n2, n3, n4 in the correct orientation (supposedly).

		////std::cout<<"GROWTH ERROR 6"<<std::endl;	
		int edgeindex, a, a1, a2, a3, temp1, temp2;
		//std::cout<<"maxNodeCount = "<< generalParams.maxNodeCount<<std::endl;
		double newx = (coordInfoVecs.nodeLocX[coordInfoVecs.edges2Nodes_1[iedge]] + coordInfoVecs.nodeLocX[coordInfoVecs.edges2Nodes_2[iedge]])/2.0;
		////std::cout<<"GROWTH ERROR 6.1"<<std::endl;	
		coordInfoVecs.nodeLocX[generalParams. maxNodeCount] = newx;
		////std::cout<<"GROWTH ERROR 6.2"<<std::endl;	
		double newy = (coordInfoVecs.nodeLocY[coordInfoVecs.edges2Nodes_1[iedge]] + coordInfoVecs.nodeLocY[coordInfoVecs.edges2Nodes_2[iedge]])/2.0;
		////std::cout<<"GROWTH ERROR 6.3"<<std::endl;	
		coordInfoVecs.nodeLocY[generalParams. maxNodeCount] = newy;
		////std::cout<<"GROWTH ERROR 6.4"<<std::endl;	
		double newz = (coordInfoVecs.nodeLocZ[coordInfoVecs.edges2Nodes_1[iedge]] + coordInfoVecs.nodeLocZ[coordInfoVecs.edges2Nodes_2[iedge]])/2.0;
		////std::cout<<"GROWTH ERROR 6.5"<<std::endl;	
		coordInfoVecs.nodeLocZ[generalParams. maxNodeCount] = newz;
		//These are the coordinate of the new vertex. Its index is "coordInfoVecs.nodeLocX.size()-1"

		//Before editing major data structures, we will update the nndata here since it is only affected by the addition of new nodes.

		//int NODESIZE= generalParams.maxNodeCount;//coordInfoVecs.nodeLocX.size();
		////std::cout<<"GROWTH ERROR 7"<<std::endl;			
		coordInfoVecs.triangles2Nodes_1[coordInfoVecs.num_triangles] = n1;
		coordInfoVecs.triangles2Nodes_2[coordInfoVecs.num_triangles] = n2;
		coordInfoVecs.triangles2Nodes_3[coordInfoVecs.num_triangles] = generalParams.maxNodeCount;
		coordInfoVecs.num_triangles += 1;
		//NOTE: What this +1 actually does is that it specifies the location to write
		//any new data. Here it points to the location to write new triangles information.
		//This is a new triangle associated with (tn1, tn2, newnode). Its index is "coordInfoVecs.triangles2Nodes_1.size()-4".
		////std::cout<<"GROWTH ERROR 8"<<std::endl;	
		coordInfoVecs.triangles2Nodes_1[coordInfoVecs.num_triangles] =(n2);
		coordInfoVecs.triangles2Nodes_2[coordInfoVecs.num_triangles] =(n3);
		coordInfoVecs.triangles2Nodes_3[coordInfoVecs.num_triangles] = generalParams.maxNodeCount;
		coordInfoVecs.num_triangles += 1;
		//This is a new triangle associated with (tn2, tn3, newnode). Its index is "coordInfoVecs.triangles2Nodes_1.size()-3".
		////std::cout<<"GROWTH ERROR 9"<<std::endl;	
		coordInfoVecs.triangles2Nodes_1[coordInfoVecs.num_triangles] =(n3);
		coordInfoVecs.triangles2Nodes_2[coordInfoVecs.num_triangles] =(n4);
		coordInfoVecs.triangles2Nodes_3[coordInfoVecs.num_triangles] = generalParams.maxNodeCount;
		coordInfoVecs.num_triangles += 1;
		//This is a new triangle associated with (tn3, tn1, newnode). Its index is "coordInfoVecs.triangles2Nodes_1.size()-2".
		////std::cout<<"GROWTH ERROR 10"<<std::endl;	
		coordInfoVecs.triangles2Nodes_1[coordInfoVecs.num_triangles] =(n4);
		coordInfoVecs.triangles2Nodes_2[coordInfoVecs.num_triangles] =(n1);
		coordInfoVecs.triangles2Nodes_3[coordInfoVecs.num_triangles] = generalParams.maxNodeCount;
		coordInfoVecs.num_triangles += 1;
		//This is a new triangle associated with (tn3, tn1, newnode). Its index is "coordInfoVecs.triangles2Nodes_1.size()-1".
		////std::cout<<"GROWTH ERROR 11"<<std::endl;	
		//Now we add new edges formed by the addition of the new node.
		coordInfoVecs.edges2Nodes_1[coordInfoVecs.num_edges] = (generalParams.maxNodeCount);
		coordInfoVecs.edges2Nodes_2[coordInfoVecs.num_edges] = (n1);
		coordInfoVecs.edges2Triangles_1[coordInfoVecs.num_edges] = coordInfoVecs.num_triangles - 4;
		coordInfoVecs.edges2Triangles_2[coordInfoVecs.num_edges] = coordInfoVecs.num_triangles - 1;
		coordInfoVecs.num_edges += 1;
		////std::cout<<"GROWTH ERROR 12"<<std::endl;	
		//This is a new edge associated with (newnode, tn1). Its index is "edges2Nodes_1.size()-4".
		coordInfoVecs.edges2Nodes_1[coordInfoVecs.num_edges] = (generalParams.maxNodeCount);
		coordInfoVecs.edges2Nodes_2[coordInfoVecs.num_edges] = (n2);
		coordInfoVecs.edges2Triangles_1[coordInfoVecs.num_edges] = coordInfoVecs.num_triangles - 3;
		coordInfoVecs.edges2Triangles_2[coordInfoVecs.num_edges] = coordInfoVecs.num_triangles - 4;
		coordInfoVecs.num_edges += 1;
		////std::cout<<"GROWTH ERROR 13"<<std::endl;	
		//This is a new edge associated with (newnode, tn2). Its index is "edges2Nodes_1.size()-3".
		coordInfoVecs.edges2Nodes_1[coordInfoVecs.num_edges] = (generalParams.maxNodeCount);
		coordInfoVecs.edges2Nodes_2[coordInfoVecs.num_edges] = (n3);
		coordInfoVecs.edges2Triangles_1[coordInfoVecs.num_edges] = coordInfoVecs.num_triangles - 2;
		coordInfoVecs.edges2Triangles_2[coordInfoVecs.num_edges] = coordInfoVecs.num_triangles - 3;
		coordInfoVecs.num_edges += 1;
		////std::cout<<"GROWTH ERROR 14"<<std::endl;	
		//This is a new edge associated with (newnode, tn3). Its index is "edges2Nodes_1.size()-2".
		coordInfoVecs.edges2Nodes_1[coordInfoVecs.num_edges] = (generalParams.maxNodeCount);
		coordInfoVecs.edges2Nodes_2[coordInfoVecs.num_edges] = (n4);
		coordInfoVecs.edges2Triangles_1[coordInfoVecs.num_edges] = coordInfoVecs.num_triangles - 1;
		coordInfoVecs.edges2Triangles_2[coordInfoVecs.num_edges] = coordInfoVecs.num_triangles - 2;
		coordInfoVecs.num_edges += 1;
		////std::cout<<"GROWTH ERROR 15"<<std::endl;	
		for (int j = 0; j < 4; j++){
		//	//std::cout<<"GROWTH ERROR 16"<<std::endl;				
			//Now we check to see if the order of update is correct, i.e. are edges2Triangles data in correct orientation.
			//This is crucial in the bendingspring computation.
			edgeindex = (coordInfoVecs.num_edges - (4-j));
			a = coordInfoVecs.edges2Triangles_1[edgeindex];
			if ((coordInfoVecs.triangles2Nodes_1[a] == coordInfoVecs.edges2Nodes_1[edgeindex]) && (coordInfoVecs.triangles2Nodes_2[a] == coordInfoVecs.edges2Nodes_2[edgeindex])){
				a1 = 1;
			}
			else{
				a1 = 0;
			}
			if ((coordInfoVecs.triangles2Nodes_2[a] == coordInfoVecs.edges2Nodes_1[edgeindex]) && (coordInfoVecs.triangles2Nodes_3[a] == coordInfoVecs.edges2Nodes_2[edgeindex])){
				a2 = 1;
			}
			else{
				a2 = 0;
			}
			if ((coordInfoVecs.triangles2Nodes_3[a] == coordInfoVecs.edges2Nodes_1[edgeindex]) && (coordInfoVecs.triangles2Nodes_1[a] == coordInfoVecs.edges2Nodes_2[edgeindex])){
				a3 = 1;
			}
			else{
				a3 = 0;
			}

			if ((a1+a2+a3) == 0){
				temp1 = coordInfoVecs.edges2Triangles_1[edgeindex];
				temp2 = coordInfoVecs.edges2Triangles_2[edgeindex];
				coordInfoVecs.edges2Triangles_1[edgeindex] = temp2;
				coordInfoVecs.edges2Triangles_2[edgeindex] = temp1;
			}
			else{}
			//This checks if the orientation is correct or not, if not, flip the ordering.
		}
		//This is a new edge associated with (newnode, tn3). Its index is "edges2Nodes_1.size()-1".
		generalParams.maxNodeCount += 1;

		coordInfoVecs.nndata1[generalParams.maxNodeCount-1] =  (n1);
		coordInfoVecs.nndata2[generalParams.maxNodeCount-1] =  (n2);
		coordInfoVecs.nndata3[generalParams.maxNodeCount-1] =  (n3);
		coordInfoVecs.nndata4[generalParams.maxNodeCount-1] =  (n4);
		coordInfoVecs.nndata5[generalParams.maxNodeCount-1] =  (-2);
		coordInfoVecs.nndata6[generalParams.maxNodeCount-1] =  (-2);
		coordInfoVecs.nndata7[generalParams.maxNodeCount-1] =  (-2);
		coordInfoVecs.nndata8[generalParams.maxNodeCount-1] =  (-2);
		coordInfoVecs.nndata9[generalParams.maxNodeCount-1] =  (-2);
		coordInfoVecs.nndata10[generalParams.maxNodeCount-1] = (-2);
		coordInfoVecs.nndata11[generalParams.maxNodeCount-1] = (-2);
		coordInfoVecs.nndata12[generalParams.maxNodeCount-1] = (-2);
		for (int j = 0; j < 2; j++){
			int nn, nnn, nnnn;
			if (j == 0){
				nn = n1;
				nnn = n3;
				nnnn = generalParams.maxNodeCount-1;
			}
			else if (j == 1){
				nn = n3;
				nnn = n1;
				nnnn = generalParams.maxNodeCount-1;
			}
			if (coordInfoVecs.nndata1[nn] == nnn){
				coordInfoVecs.nndata1[nn] = nnnn;
			}
			else if (coordInfoVecs.nndata2[nn] == nnn){
				coordInfoVecs.nndata2[nn] = nnnn;
			}
			else if (coordInfoVecs.nndata3[nn] == nnn){
				coordInfoVecs.nndata3[nn] = nnnn;
			}
			else if (coordInfoVecs.nndata4[nn] == nnn){
				coordInfoVecs.nndata4[nn] = nnnn;
			}
			else if (coordInfoVecs.nndata5[nn] == nnn){
				coordInfoVecs.nndata5[nn] = nnnn;
			}
			else if (coordInfoVecs.nndata6[nn] == nnn){
				coordInfoVecs.nndata6[nn] = nnnn;
			}
			else if (coordInfoVecs.nndata7[nn] == nnn){
				coordInfoVecs.nndata7[nn] = nnnn;
			}
			else if (coordInfoVecs.nndata8[nn] == nnn){
				coordInfoVecs.nndata8[nn] = nnnn;
			}
			else if (coordInfoVecs.nndata9[nn] == nnn){
				coordInfoVecs.nndata9[nn] = nnnn;
			}
			else if (coordInfoVecs.nndata10[nn] == nnn){
				coordInfoVecs.nndata10[nn] = nnnn;
			}
			else if (coordInfoVecs.nndata11[nn] == nnn){
				coordInfoVecs.nndata11[nn] = nnnn;
			}
			else if (coordInfoVecs.nndata12[nn] == nnn){
				coordInfoVecs.nndata12[nn] = nnnn;
			}
		}

		for (int j = 0; j < 2; j++){
			int nn, nnn;
			if (j == 0){
				nn = n2;
				nnn = generalParams.maxNodeCount-1;
			}
			else if (j == 1){
				nn = n4;
				nnn = generalParams.maxNodeCount-1;
			}
			if (coordInfoVecs.nndata1[nn] < 0){
				coordInfoVecs.nndata1[nn] = nnn;
			}
			else if (coordInfoVecs.nndata2[nn] < 0){
				coordInfoVecs.nndata2[nn] = nnn;
			}
			else if (coordInfoVecs.nndata3[nn] < 0){
				coordInfoVecs.nndata3[nn] = nnn;
			}
			else if (coordInfoVecs.nndata4[nn] < 0){
				coordInfoVecs.nndata4[nn] = nnn;
			}
			else if (coordInfoVecs.nndata5[nn] < 0){
				coordInfoVecs.nndata5[nn] = nnn;
			}
			else if (coordInfoVecs.nndata6[nn] < 0){
				coordInfoVecs.nndata6[nn] = nnn;
			}
			else if (coordInfoVecs.nndata7[nn] < 0){
				coordInfoVecs.nndata7[nn] = nnn;
			}
			else if (coordInfoVecs.nndata8[nn] < 0){
				coordInfoVecs.nndata8[nn] = nnn;
			}
			else if (coordInfoVecs.nndata9[nn] < 0){
				coordInfoVecs.nndata9[nn] = nnn;
			}
			else if (coordInfoVecs.nndata10[nn] < 0){
				coordInfoVecs.nndata10[nn] = nnn;
			}
			else if (coordInfoVecs.nndata11[nn] < 0){
				coordInfoVecs.nndata11[nn] = nnn;
			}
			else if (coordInfoVecs.nndata12[nn] < 0){
				coordInfoVecs.nndata12[nn] = nnn;
			}
		}
		//generalParams.num_of_nodes += 1;

		


		////std::cout<<"GROWTH ERROR 17"<<std::endl;	
		//Now we update the edges2Triangles data structure with new edges.
		//std::cout<<"elem 1 = "<<elem1<<std::endl;
		//std::cout<<"elem 2 = "<<elem2<<std::endl;
		for (int i = 0; i < coordInfoVecs.num_edges; i++){
		//	std::cout<<"edges2triangles"<<" "<< i <<" : "<<coordInfoVecs. edges2Triangles_1[i]<<" "<<coordInfoVecs. edges2Triangles_2[i]<<std::endl;
		}
		int TRIANGLESIZE = coordInfoVecs.num_triangles;//coordInfoVecs.triangles2Nodes_1.size();
		if (coordInfoVecs.edges2Triangles_1[t1e1] == elem1){
			coordInfoVecs.edges2Triangles_1[t1e1] = TRIANGLESIZE-4;
		}
		else if (coordInfoVecs.edges2Triangles_2[t1e1] == elem1){
			coordInfoVecs.edges2Triangles_2[t1e1] = TRIANGLESIZE-4;
		}
		else{}
		////std::cout<<"GROWTH ERROR 18"<<std::endl;	
		if (coordInfoVecs.edges2Triangles_1[t1e2] == elem1){
			coordInfoVecs.edges2Triangles_1[t1e2] = TRIANGLESIZE-3;
		}
		else if (coordInfoVecs.edges2Triangles_2[t1e2] == elem1){
			coordInfoVecs.edges2Triangles_2[t1e2] = TRIANGLESIZE-3;
		}
		else{}
		////std::cout<<"GROWTH ERROR 19"<<std::endl;	
		if (coordInfoVecs.edges2Triangles_1[t2e1] == elem2){
			coordInfoVecs.edges2Triangles_1[t2e1] = TRIANGLESIZE-2;
		}
		else if (coordInfoVecs.edges2Triangles_2[t2e1] == elem2){
			coordInfoVecs.edges2Triangles_2[t2e1] = TRIANGLESIZE-2;
		}
		else{}
		////std::cout<<"GROWTH ERROR 20"<<std::endl;	
		if (coordInfoVecs.edges2Triangles_1[t2e2] == elem2){
			coordInfoVecs.edges2Triangles_1[t2e2] = TRIANGLESIZE-1;
		}
		else if (coordInfoVecs.edges2Triangles_2[t2e2] == elem2){
			coordInfoVecs.edges2Triangles_2[t2e2] = TRIANGLESIZE-1;
		}
		else{}
		//std::cout<<"t1e1 "<<t1e1<<std::endl;
		//std::cout<<"t1e2 "<<t1e2<<std::endl;
		//std::cout<<"t1e3 "<<t1e3<<std::endl;
		//std::cout<<"t2e1 "<<t2e1<<std::endl;
		//std::cout<<"t2e2 "<<t2e2<<std::endl;
		//std::cout<<"t2e3 "<<t2e3<<std::endl;

		//for (int i = 0; i < coordInfoVecs.num_edges; i++){
		//	std::cout<<"edges2triangles"<<" "<< i <<" : "<<coordInfoVecs. edges2Triangles_1[i]<<" "<<coordInfoVecs. edges2Triangles_2[i]<<std::endl;
		//}
		//The above change the existing edges2Triangles data structure to accomodate new triangles added.

		//////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
		//////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
		

		//////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
		//////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
		
		//Now we will take care of the last unedited data structure "triangles2Edges".
		//int aa, bb;
		int EDGESIZE = coordInfoVecs.num_edges;//coordInfoVecs.edges2Nodes_1.size();
		for (int j = 0; j < 4; j++){
		//	//std::cout<<"GROWTH ERROR 21"<<std::endl;	
			if (j == 0){
				coordInfoVecs.triangles2Edges_1[coordInfoVecs.num_triangles - 4] = (EDGESIZE-4);
				coordInfoVecs.triangles2Edges_2[coordInfoVecs.num_triangles - 4] = (t1e1);
				coordInfoVecs.triangles2Edges_3[coordInfoVecs.num_triangles - 4] = (EDGESIZE-3);   
			}
			else if (j == 1){
				coordInfoVecs.triangles2Edges_1[coordInfoVecs.num_triangles - 3] = (EDGESIZE-3);
				coordInfoVecs.triangles2Edges_2[coordInfoVecs.num_triangles - 3] = (t1e2);
				coordInfoVecs.triangles2Edges_3[coordInfoVecs.num_triangles - 3] = (EDGESIZE-2);   
			}
			else if (j ==2){
				coordInfoVecs.triangles2Edges_1[coordInfoVecs.num_triangles - 2] = (EDGESIZE-2);
				coordInfoVecs.triangles2Edges_2[coordInfoVecs.num_triangles - 2] = (t2e1);
				coordInfoVecs.triangles2Edges_3[coordInfoVecs.num_triangles - 2] = (EDGESIZE-1);   
			}
			else if (j ==3){
				coordInfoVecs.triangles2Edges_1[coordInfoVecs.num_triangles - 1] = (EDGESIZE-1);
				coordInfoVecs.triangles2Edges_2[coordInfoVecs.num_triangles - 1] = (t2e2);
				coordInfoVecs.triangles2Edges_3[coordInfoVecs.num_triangles - 1] = (EDGESIZE-4);   
			}
			
		}
	
		
		if (generalParams.nodes_in_upperhem[coordInfoVecs.edges2Nodes_1[iedge]] == 1 && generalParams.nodes_in_upperhem[coordInfoVecs.edges2Nodes_2[iedge]] == 1){
			generalParams.nodes_in_upperhem.push_back(1);
		}
		else{
			generalParams.nodes_in_upperhem.push_back(-1);
		}
		//Finally, we will fill the edge data chosen for growth (expansion) with INT_MAX so its data is no longer relevant to the computation
		////std::cout<<"GROWTH ERROR 22"<<std::endl;	
		coordInfoVecs.edges2Nodes_1[iedge] = INT_MAX;
		coordInfoVecs.edges2Nodes_2[iedge] = INT_MAX;
		for (int i = 0; i < coordInfoVecs.num_triangles; i++){
		//	//std::cout<<"GROWTH ERROR 23"<<std::endl;	
			if (coordInfoVecs.triangles2Edges_1[i] == iedge){
				coordInfoVecs.triangles2Edges_1[i] = INT_MAX;
			}
			if (coordInfoVecs.triangles2Edges_2[i] == iedge){
				coordInfoVecs.triangles2Edges_2[i] = INT_MAX;
			}
			if (coordInfoVecs.triangles2Edges_3[i] == iedge){
				coordInfoVecs.triangles2Edges_3[i] = INT_MAX;
			}
		}
		coordInfoVecs.edges2Triangles_1[iedge] = INT_MAX;
		coordInfoVecs.edges2Triangles_2[iedge] = INT_MAX;
		
		////std::cout<<"GROWTH ERROR 24"<<std::endl;	
		
			coordInfoVecs.triangles2Nodes_1[elem1] = INT_MAX;
			coordInfoVecs.triangles2Nodes_2[elem1] = INT_MAX;
			coordInfoVecs.triangles2Nodes_3[elem1] = INT_MAX;
			coordInfoVecs.triangles2Nodes_1[elem2] = INT_MAX;
			coordInfoVecs.triangles2Nodes_2[elem2] = INT_MAX;
			coordInfoVecs.triangles2Nodes_3[elem2] = INT_MAX;
			
			//Delete the associated vertices information of the selected triangle.
			//Since we delete the chosen triangles, any triangle indexed lower than the deleted one will have its index reduced (or moved up) by 1.
			//Hence, we need to sweep through all data structures using the triangle index to change the index accordingly.
			for (int i = 0; i < coordInfoVecs.num_edges; i++){
		//		//std::cout<<"GROWTH ERROR 25"<<std::endl;	
				if (coordInfoVecs.edges2Triangles_1[i] == elem1 || coordInfoVecs.edges2Triangles_1[i] == elem2){
					coordInfoVecs.edges2Triangles_1[i] = INT_MAX;
				}
				if (coordInfoVecs.edges2Triangles_2[i] == elem1 || coordInfoVecs.edges2Triangles_2[i] == elem2){
					coordInfoVecs.edges2Triangles_2[i] = INT_MAX;
				}
			if (coordInfoVecs.edges2Triangles_1[i] != INT_MAX && coordInfoVecs.edges2Triangles_2[i] == INT_MAX){
				std::cout<<"modified edges2Triangles "<<coordInfoVecs.edges2Triangles_1[i]<<" "<<coordInfoVecs.edges2Triangles_2[i]<<std::endl;
				}
				else if (coordInfoVecs.edges2Triangles_1[i] == INT_MAX && coordInfoVecs.edges2Triangles_2[i] != INT_MAX){
					std::cout<<"modified edges2Triangles "<<coordInfoVecs.edges2Triangles_1[i]<<" "<<coordInfoVecs.edges2Triangles_2[i]<<std::endl;
					}
			}
			//This completes the sweep. After this, the indices of triangle used in edges2Triangles data structure should be the correct one.
		//	//std::cout<<"GROWTH ERROR 26"<<std::endl;	
			coordInfoVecs.triangles2Edges_1[elem1] = INT_MAX;
			coordInfoVecs.triangles2Edges_2[elem1] = INT_MAX;
			coordInfoVecs.triangles2Edges_3[elem1] = INT_MAX;
			coordInfoVecs.triangles2Edges_1[elem2] = INT_MAX;
			coordInfoVecs.triangles2Edges_2[elem2] = INT_MAX;
			coordInfoVecs.triangles2Edges_3[elem2] = INT_MAX;
			for (int i = 0; i < coordInfoVecs.num_triangles; i++){
		//		//std::cout<<"GROWTH ERROR 27"<<std::endl;	
				if (coordInfoVecs.triangles2Edges_1[i] == iedge){
					coordInfoVecs.triangles2Edges_1[i] = INT_MAX;
				}
				if (coordInfoVecs.triangles2Edges_2[i] == iedge ){
					coordInfoVecs.triangles2Edges_2[i] = INT_MAX;
				}
				if (coordInfoVecs.triangles2Edges_3[i] == iedge ){
					coordInfoVecs.triangles2Edges_3[i] = INT_MAX;
				}
			}
		
		//Erase the edge infomation related to the deleted triangle. Note the deletion should always start with the largest index.

		//Before we delete the edge, determine whether the newly added node is part of nodes_in_upperhem or not.
		

		
						//Erase the edge infomation related to the deleted triangle.

						//Now we update the nodes_in_upperhem and edges_in_upperhem data structures.
						//This ensures that the newly created edges will have the correct associated spring constant.
//std::cout<<"ERROR HERE?"<<std::endl;
		//generalParams.edges_in_upperhem[iedge] = INT_MAX;
		for (int i = 0; i < edges_in_upperhem.size(); i++){
			if (edges_in_upperhem[i] == iedge){
				edges_in_upperhem[i] == INT_MAX;
				//break;
			}
		}

		for (int q = 0; q < 4; q++){
		//	//std::cout<<"GROWTH ERROR 30"<<std::endl;	
			int nodeP = coordInfoVecs.triangles2Nodes_1[coordInfoVecs.num_triangles - (4-q)]; 
			int nodeQ = coordInfoVecs.triangles2Nodes_2[coordInfoVecs.num_triangles - (4-q)];
			int nodeR = coordInfoVecs.triangles2Nodes_3[coordInfoVecs.num_triangles - (4-q)];

			if (generalParams.nodes_in_upperhem[nodeP]==1 && generalParams.nodes_in_upperhem[nodeQ] ==1 && generalParams.nodes_in_upperhem[nodeR] ==1){
				generalParams.triangles_in_upperhem.push_back(1);
			}
			else if (generalParams.nodes_in_upperhem[nodeP]==1 && generalParams.nodes_in_upperhem[nodeQ] ==1){
				generalParams.triangles_in_upperhem.push_back(0);
			}
			else if (generalParams.nodes_in_upperhem[nodeP]==1 && generalParams.nodes_in_upperhem[nodeR] ==1){
				generalParams.triangles_in_upperhem.push_back(0);
			}
			else if (generalParams.nodes_in_upperhem[nodeQ] ==1 && generalParams.nodes_in_upperhem[nodeR] ==1){
				generalParams.triangles_in_upperhem.push_back(0);
			}
			else{
				generalParams.triangles_in_upperhem.push_back(INT_MAX);
			}
		}
		//std::cout<<"edges2Triangles size"<<""<<coordInfoVecs.edges2Triangles_1.size()<<" "<<coordInfoVecs.edges2Triangles_2.size()<<std::endl;
		//std::cout<<"triangles_in_upperhem size "<<generalParams.triangles_in_upperhem.size()<<std::endl;	
		//std::cout<<"GROWTH ERROR 29"<<std::endl;	
		for (int q = 0; q < 4; q++){
			//std::cout<<"GROWTH ERROR 31"<<std::endl;	
			int elem_1 = coordInfoVecs.edges2Triangles_1[coordInfoVecs.num_edges-(4 - q)];
			//std::cout<<coordInfoVecs.num_edges-(4 - q)<<std::endl;
			//std::cout<<"elem_1 "<<elem_1<<std::endl;
			//std::cout<<generalParams.nodes_in_upperhem[nodeP]<<std::endl;
			int elem_2 = coordInfoVecs.edges2Triangles_2[coordInfoVecs.num_edges-(4 - q)];
			//std::cout<<"elem_2"<<elem_2<<std::endl;
			//std::cout<<generalParams.nodes_in_upperhem[nodeQ]<<std::endl;
			//std::cout<<"GROWTH ERROR 31.5"<<std::endl;
			if (generalParams.triangles_in_upperhem[elem_1] == 1 && generalParams.triangles_in_upperhem[elem_2] == 1){
				
				generalParams.edges_in_upperhem.push_back(1);
				//generalParams.edges_in_upperhem_index.push_back(generalParams.num_of_edges - (4 - q));
				edges_in_upperhem.push_back(coordInfoVecs.num_edges - (4 - q));
			}
			
			else if (generalParams.triangles_in_upperhem[elem_1] == 1 || generalParams.triangles_in_upperhem[elem_2] == 1){
				
				generalParams.edges_in_upperhem.push_back(0);
				//generalParams.edges_in_upperhem_index.push_back(generalParams.num_of_edges - (4 - q));
			}
			else{
				
				generalParams.edges_in_upperhem.push_back(INT_MAX);
			}

			generalParams.boundaries_in_upperhem.push_back(-1);
			
			
		}
		generalParams.triangles_in_upperhem[elem1] = INT_MAX;
		generalParams.triangles_in_upperhem[elem2] = INT_MAX;
		
						
						

						//Here we regenerate the edges that we will loop through in both edgeswap and growth, if any edge deletion actually happened.
						
						//if (triggered_counter > 0){
						//	edges_in_upperhem_for_loop.clear();
						//	for (int i = 0; i < generalParams.edges_in_upperhem_index.size(); i++){
						//		if (generalParams.edges_in_upperhem_index[i] != INT_MAX){
						//			edges_in_upperhem_for_loop.push_back(generalParams.edges_in_upperhem_index[i]);
						//		}
						//	}
						//}
						
						//This should completes the dreadful data structure update associated with cell (membrane) growth.
						//Have fun modifying it if you need more function!
						//if (triggered == true){
						//	break;
						//}
					}

//std::cout<<"GROWTH DONE!"<<std::endl;
 ////storage->print_VTK_File();
////storage->storeVariables();

/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
//////////////////////////////////////////////////// END OF GROWTH SECTION //////////////////////////////////////////////////////////////////////////
/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

 					
					
 			}
		
		}
		

	};
	
	





void System::assignStorage(std::shared_ptr<Storage> _storage) {
	storage = _storage;
};
void System::set_weak_builder(std::weak_ptr<SystemBuilder> _weak_bld_ptr) {
	weak_bld_ptr = _weak_bld_ptr;
};



//initialize memory for thrust vectors and set coordInfoVecs vals from input. 
void System::initializeSystem(HostSetInfoVecs& hostSetInfoVecs) {
	std::cout<<"Initializing"<<std::endl;

	generalParams.maxNodeCount = hostSetInfoVecs.nodeLocX.size();
	coordInfoVecs.num_edges = hostSetInfoVecs.edges2Nodes_1.size();
	coordInfoVecs.num_triangles = hostSetInfoVecs.triangles2Nodes_1.size();

	std::cout<<"num nodes: "<< generalParams.maxNodeCount << std::endl;
	std::cout<<"num edges: "<< coordInfoVecs.num_edges << std::endl;
	std::cout<<"num elems: "<< coordInfoVecs.num_triangles << std::endl;
	//allocate memory
	int mem_prealloc = 4;
	coordInfoVecs.isNodeFixed.resize(mem_prealloc*hostSetInfoVecs.nodeLocX.size());
	coordInfoVecs.prevNodeLocX.resize(mem_prealloc*hostSetInfoVecs.nodeLocX.size());
	coordInfoVecs.prevNodeLocY.resize(mem_prealloc*hostSetInfoVecs.nodeLocX.size());
	coordInfoVecs.prevNodeLocZ.resize(mem_prealloc*hostSetInfoVecs.nodeLocX.size());

	coordInfoVecs.prevNodeForceX.resize(mem_prealloc*hostSetInfoVecs.nodeLocX.size());
	coordInfoVecs.prevNodeForceY.resize(mem_prealloc*hostSetInfoVecs.nodeLocX.size());
	coordInfoVecs.prevNodeForceZ.resize(mem_prealloc*hostSetInfoVecs.nodeLocX.size());
	
	coordInfoVecs.nodeLocX.resize(mem_prealloc*hostSetInfoVecs.nodeLocX.size());
	coordInfoVecs.nodeLocY.resize(mem_prealloc*hostSetInfoVecs.nodeLocX.size());
	coordInfoVecs.nodeLocZ.resize(mem_prealloc*hostSetInfoVecs.nodeLocX.size());

	coordInfoVecs.nodeForceX.resize(mem_prealloc*hostSetInfoVecs.nodeLocX.size(), 0.0);
	coordInfoVecs.nodeForceY.resize(mem_prealloc*hostSetInfoVecs.nodeLocX.size(), 0.0);
	coordInfoVecs.nodeForceZ.resize(mem_prealloc*hostSetInfoVecs.nodeLocX.size(), 0.0);

	coordInfoVecs.triangles2Nodes_1.resize( mem_prealloc*coordInfoVecs.num_triangles );
	coordInfoVecs.triangles2Nodes_2.resize( mem_prealloc*coordInfoVecs.num_triangles );
	coordInfoVecs.triangles2Nodes_3.resize( mem_prealloc*coordInfoVecs.num_triangles );
	
	coordInfoVecs.triangles2Edges_1.resize( mem_prealloc*coordInfoVecs.num_triangles );
	coordInfoVecs.triangles2Edges_2.resize( mem_prealloc*coordInfoVecs.num_triangles );
	coordInfoVecs.triangles2Edges_3.resize( mem_prealloc*coordInfoVecs.num_triangles );

	coordInfoVecs.edges2Nodes_1.resize( mem_prealloc*coordInfoVecs.num_edges );
	coordInfoVecs.edges2Nodes_2.resize( mem_prealloc*coordInfoVecs.num_edges );
	
	coordInfoVecs.edges2Triangles_1.resize( mem_prealloc*coordInfoVecs.num_edges );
	coordInfoVecs.edges2Triangles_2.resize( mem_prealloc*coordInfoVecs.num_edges );

	coordInfoVecs.nndata1.resize( mem_prealloc*generalParams.maxNodeCount);
	coordInfoVecs.nndata2.resize( mem_prealloc*generalParams.maxNodeCount);
	coordInfoVecs.nndata3.resize( mem_prealloc*generalParams.maxNodeCount);
	coordInfoVecs.nndata4.resize( mem_prealloc*generalParams.maxNodeCount);
	coordInfoVecs.nndata5.resize( mem_prealloc*generalParams.maxNodeCount);
	coordInfoVecs.nndata6.resize( mem_prealloc*generalParams.maxNodeCount);
	coordInfoVecs.nndata7.resize( mem_prealloc*generalParams.maxNodeCount);
	coordInfoVecs.nndata8.resize( mem_prealloc*generalParams.maxNodeCount);
	coordInfoVecs.nndata9.resize( mem_prealloc*generalParams.maxNodeCount);
	coordInfoVecs.nndata10.resize( mem_prealloc*generalParams.maxNodeCount);
	coordInfoVecs.nndata11.resize( mem_prealloc*generalParams.maxNodeCount);
	coordInfoVecs.nndata12.resize( mem_prealloc*generalParams.maxNodeCount);



	//copy info to GPU
	std::cout<<"Copying"<<std::endl;
	thrust::copy(hostSetInfoVecs.isNodeFixed.begin(),hostSetInfoVecs.isNodeFixed.end(), coordInfoVecs.isNodeFixed.begin());
	
	std::cout<<"fixed_node_in_host: "<<std::endl;
	for (int k = 0; k < hostSetInfoVecs.isNodeFixed.size(); k++){
		//std::cout<<hostSetInfoVecs.isNodeFixed[k]<<std::endl;
	}
	std::cout<<"end_of_fixed_node_host_printout"<<std::endl;
	std::cout<<"fixed_node_in_device: "<<std::endl;
	for (int k = 0; k < coordInfoVecs.isNodeFixed.size(); k++){
		//std::cout<<coordInfoVecs.isNodeFixed[k]<<std::endl;
	}
	std::cout<<"end_of_fixed_node_device_printout"<<std::endl;
std::cout<<"size of host fixed "<< hostSetInfoVecs.isNodeFixed.size()<<std::endl;
std::cout<<"size of device fixed "<< coordInfoVecs.isNodeFixed.size()<<std::endl;

	/*for (int k = 0; k < coordInfoVecs.isNodeFixed.size(); k++){
		bool isFixedHost = hostSetInfoVecs.isNodeFixed[k];
		bool isFixedDevice = coordInfoVecs.isNodeFixed[k];
		if (isFixedDevice != isFixedHost){

			std::cout<<"pos "<< k << " dev val = " << coordInfoVecs.isNodeFixed[k]
				<< " host val = " <<  hostSetInfoVecs.isNodeFixed[k] <<std::endl;
		}
	}*/
	thrust::fill(coordInfoVecs.nodeForceX.begin(), coordInfoVecs.nodeForceX.end(), 0.0);
	thrust::fill(coordInfoVecs.nodeForceY.begin(), coordInfoVecs.nodeForceY.end(), 0.0);
	thrust::fill(coordInfoVecs.nodeForceZ.begin(), coordInfoVecs.nodeForceZ.end(), 0.0);

	thrust::fill(coordInfoVecs.prevNodeForceX.begin(), coordInfoVecs.prevNodeForceX.end(), 0.0);
	thrust::fill(coordInfoVecs.prevNodeForceY.begin(), coordInfoVecs.prevNodeForceY.end(), 0.0);
	thrust::fill(coordInfoVecs.prevNodeForceZ.begin(), coordInfoVecs.prevNodeForceZ.end(), 0.0);
	
	thrust::copy(hostSetInfoVecs.nodeLocX.begin(), hostSetInfoVecs.nodeLocX.end(), coordInfoVecs.prevNodeLocX.begin() );
	thrust::copy(hostSetInfoVecs.nodeLocY.begin(), hostSetInfoVecs.nodeLocY.end(), coordInfoVecs.prevNodeLocY.begin() );
	thrust::copy(hostSetInfoVecs.nodeLocZ.begin(), hostSetInfoVecs.nodeLocZ.end(), coordInfoVecs.prevNodeLocZ.begin() );
	
	thrust::copy(hostSetInfoVecs.nodeLocX.begin(), hostSetInfoVecs.nodeLocX.end(), coordInfoVecs.nodeLocX.begin() );
	thrust::copy(hostSetInfoVecs.nodeLocY.begin(), hostSetInfoVecs.nodeLocY.end(), coordInfoVecs.nodeLocY.begin() );
	thrust::copy(hostSetInfoVecs.nodeLocZ.begin(), hostSetInfoVecs.nodeLocZ.end(), coordInfoVecs.nodeLocZ.begin() );
	
	thrust::copy(hostSetInfoVecs.triangles2Nodes_1.begin(), hostSetInfoVecs.triangles2Nodes_1.end(), coordInfoVecs.triangles2Nodes_1.begin() );
	thrust::copy(hostSetInfoVecs.triangles2Nodes_2.begin(), hostSetInfoVecs.triangles2Nodes_2.end(), coordInfoVecs.triangles2Nodes_2.begin() );
	thrust::copy(hostSetInfoVecs.triangles2Nodes_3.begin(), hostSetInfoVecs.triangles2Nodes_3.end(), coordInfoVecs.triangles2Nodes_3.begin() );
	
	thrust::copy(hostSetInfoVecs.triangles2Edges_1.begin(), hostSetInfoVecs.triangles2Edges_1.end(), coordInfoVecs.triangles2Edges_1.begin() );
	thrust::copy(hostSetInfoVecs.triangles2Edges_2.begin(), hostSetInfoVecs.triangles2Edges_2.end(), coordInfoVecs.triangles2Edges_2.begin() );
	thrust::copy(hostSetInfoVecs.triangles2Edges_3.begin(), hostSetInfoVecs.triangles2Edges_3.end(), coordInfoVecs.triangles2Edges_3.begin() );

	thrust::copy(hostSetInfoVecs.edges2Nodes_1.begin(), hostSetInfoVecs.edges2Nodes_1.end(), coordInfoVecs.edges2Nodes_1.begin() );
	thrust::copy(hostSetInfoVecs.edges2Nodes_2.begin(), hostSetInfoVecs.edges2Nodes_2.end(), coordInfoVecs.edges2Nodes_2.begin() );
	
	thrust::copy(hostSetInfoVecs.edges2Triangles_1.begin(), hostSetInfoVecs.edges2Triangles_1.end(), coordInfoVecs.edges2Triangles_1.begin() );
	thrust::copy(hostSetInfoVecs.edges2Triangles_2.begin(), hostSetInfoVecs.edges2Triangles_2.end(), coordInfoVecs.edges2Triangles_2.begin() );

	thrust::copy(hostSetInfoVecs.nndata1.begin(), hostSetInfoVecs.nndata1.end(), coordInfoVecs.nndata1.begin() );
	thrust::copy(hostSetInfoVecs.nndata2.begin(), hostSetInfoVecs.nndata2.end(), coordInfoVecs.nndata2.begin() );
	thrust::copy(hostSetInfoVecs.nndata3.begin(), hostSetInfoVecs.nndata3.end(), coordInfoVecs.nndata3.begin() );
	thrust::copy(hostSetInfoVecs.nndata4.begin(), hostSetInfoVecs.nndata4.end(), coordInfoVecs.nndata4.begin() );
	thrust::copy(hostSetInfoVecs.nndata5.begin(), hostSetInfoVecs.nndata5.end(), coordInfoVecs.nndata5.begin() );
	thrust::copy(hostSetInfoVecs.nndata6.begin(), hostSetInfoVecs.nndata6.end(), coordInfoVecs.nndata6.begin() );
	thrust::copy(hostSetInfoVecs.nndata7.begin(), hostSetInfoVecs.nndata7.end(), coordInfoVecs.nndata7.begin() );
	thrust::copy(hostSetInfoVecs.nndata8.begin(), hostSetInfoVecs.nndata8.end(), coordInfoVecs.nndata8.begin() );
	thrust::copy(hostSetInfoVecs.nndata9.begin(), hostSetInfoVecs.nndata9.end(), coordInfoVecs.nndata9.begin() );
	thrust::copy(hostSetInfoVecs.nndata10.begin(), hostSetInfoVecs.nndata10.end(), coordInfoVecs.nndata10.begin() );
	thrust::copy(hostSetInfoVecs.nndata11.begin(), hostSetInfoVecs.nndata11.end(), coordInfoVecs.nndata11.begin() );
	thrust::copy(hostSetInfoVecs.nndata12.begin(), hostSetInfoVecs.nndata12.end(), coordInfoVecs.nndata12.begin() );


 
	//allocate memory for other data structures.   

	//area triangle info vec
	//number of area springs is the number of triangles
	std::cout<<"Mem"<<std::endl;
	areaTriangleInfoVecs.tempNodeIdUnreduced.resize(mem_prealloc*areaTriangleInfoVecs.factor * coordInfoVecs.num_triangles);
	areaTriangleInfoVecs.tempNodeForceXUnreduced.resize(mem_prealloc*areaTriangleInfoVecs.factor * coordInfoVecs.num_triangles);
	areaTriangleInfoVecs.tempNodeForceYUnreduced.resize(mem_prealloc*areaTriangleInfoVecs.factor * coordInfoVecs.num_triangles);
	areaTriangleInfoVecs.tempNodeForceZUnreduced.resize(mem_prealloc*areaTriangleInfoVecs.factor * coordInfoVecs.num_triangles);
	
	areaTriangleInfoVecs.tempNodeIdReduced.resize(mem_prealloc*areaTriangleInfoVecs.factor * coordInfoVecs.num_triangles);
	areaTriangleInfoVecs.tempNodeForceXReduced.resize(mem_prealloc*areaTriangleInfoVecs.factor * coordInfoVecs.num_triangles);
	areaTriangleInfoVecs.tempNodeForceYReduced.resize(mem_prealloc*areaTriangleInfoVecs.factor * coordInfoVecs.num_triangles);
	areaTriangleInfoVecs.tempNodeForceZReduced.resize(mem_prealloc*areaTriangleInfoVecs.factor * coordInfoVecs.num_triangles);

	//beinding triangle info vec
	//num bending springs is the number of times each edge is between two triangles. 
	bendingTriangleInfoVecs.numBendingSprings = coordInfoVecs.num_edges;//coordInfoVecs.edges2Triangles_1.size();

	bendingTriangleInfoVecs.tempNodeIdUnreduced.resize(mem_prealloc*bendingTriangleInfoVecs.factor * bendingTriangleInfoVecs.numBendingSprings);
	bendingTriangleInfoVecs.tempNodeForceXUnreduced.resize(mem_prealloc*bendingTriangleInfoVecs.factor * bendingTriangleInfoVecs.numBendingSprings);
	bendingTriangleInfoVecs.tempNodeForceYUnreduced.resize(mem_prealloc*bendingTriangleInfoVecs.factor * bendingTriangleInfoVecs.numBendingSprings);
	bendingTriangleInfoVecs.tempNodeForceZUnreduced.resize(mem_prealloc*bendingTriangleInfoVecs.factor * bendingTriangleInfoVecs.numBendingSprings);
	
	bendingTriangleInfoVecs.tempNodeIdReduced.resize(mem_prealloc*bendingTriangleInfoVecs.factor * bendingTriangleInfoVecs.numBendingSprings);
	bendingTriangleInfoVecs.tempNodeForceXReduced.resize(mem_prealloc*bendingTriangleInfoVecs.factor * bendingTriangleInfoVecs.numBendingSprings);
	bendingTriangleInfoVecs.tempNodeForceYReduced.resize(mem_prealloc*bendingTriangleInfoVecs.factor * bendingTriangleInfoVecs.numBendingSprings);
	bendingTriangleInfoVecs.tempNodeForceZReduced.resize(mem_prealloc*bendingTriangleInfoVecs.factor * bendingTriangleInfoVecs.numBendingSprings);

	//linear springs
	
	linearSpringInfoVecs.tempNodeIdUnreduced.resize(mem_prealloc*linearSpringInfoVecs.factor*coordInfoVecs.num_edges);
	linearSpringInfoVecs.tempNodeForceXUnreduced.resize(mem_prealloc*linearSpringInfoVecs.factor*coordInfoVecs.num_edges);
	linearSpringInfoVecs.tempNodeForceYUnreduced.resize(mem_prealloc*linearSpringInfoVecs.factor*coordInfoVecs.num_edges);
	linearSpringInfoVecs.tempNodeForceZUnreduced.resize(mem_prealloc*linearSpringInfoVecs.factor*coordInfoVecs.num_edges);
	
	linearSpringInfoVecs.tempNodeIdReduced.resize(mem_prealloc*linearSpringInfoVecs.factor * coordInfoVecs.num_edges);
	linearSpringInfoVecs.tempNodeForceXReduced.resize(mem_prealloc*linearSpringInfoVecs.factor * coordInfoVecs.num_edges);
	linearSpringInfoVecs.tempNodeForceYReduced.resize(mem_prealloc*linearSpringInfoVecs.factor * coordInfoVecs.num_edges);
	linearSpringInfoVecs.tempNodeForceZReduced.resize(mem_prealloc*linearSpringInfoVecs.factor * coordInfoVecs.num_edges);
	
	linearSpringInfoVecs.edge_initial_length.resize(mem_prealloc*coordInfoVecs.num_edges,1.0);
	
	thrust::copy(hostSetInfoVecs.edge_initial_length.begin(), hostSetInfoVecs.edge_initial_length.end(), linearSpringInfoVecs.edge_initial_length.begin() );

	//Resize the hostSetInfoVecs so that we can copy data back and forth between hostSetinfoVecs and coordInfoVecs without problem.
	hostSetInfoVecs.isNodeFixed.resize(mem_prealloc*hostSetInfoVecs.nodeLocX.size());
	//hostSetInfoVecs.prevNodeLocX.resize(mem_prealloc*hostSetInfoVecs.nodeLocX.size());
	//hostSetInfoVecs.prevNodeLocY.resize(mem_prealloc*hostSetInfoVecs.nodeLocX.size());
	//hostSetInfoVecs.prevNodeLocZ.resize(mem_prealloc*hostSetInfoVecs.nodeLocX.size());

	//hostSetInfoVecs.prevNodeForceX.resize(mem_prealloc*hostSetInfoVecs.nodeLocX.size());
	//hostSetInfoVecs.prevNodeForceY.resize(mem_prealloc*hostSetInfoVecs.nodeLocX.size());
	//hostSetInfoVecs.prevNodeForceZ.resize(mem_prealloc*hostSetInfoVecs.nodeLocX.size());
	
	hostSetInfoVecs.nodeLocX.resize(coordInfoVecs.nodeLocX.size());
	hostSetInfoVecs.nodeLocY.resize(coordInfoVecs.nodeLocX.size());
	hostSetInfoVecs.nodeLocZ.resize(coordInfoVecs.nodeLocX.size());
	std::cout<<"Host_nodeLocX size = "<<hostSetInfoVecs.nodeLocX.size()<<std::endl;

	hostSetInfoVecs.nodeForceX.resize(coordInfoVecs.nodeLocX.size());
	hostSetInfoVecs.nodeForceY.resize(coordInfoVecs.nodeLocX.size());
	hostSetInfoVecs.nodeForceZ.resize(coordInfoVecs.nodeLocX.size());
	std::cout<<"Host_nodeForceX size = "<<hostSetInfoVecs.nodeLocX.size()<<std::endl;

	hostSetInfoVecs.triangles2Nodes_1.resize( coordInfoVecs.triangles2Nodes_1.size() );
	hostSetInfoVecs.triangles2Nodes_2.resize( coordInfoVecs.triangles2Nodes_2.size() );
	hostSetInfoVecs.triangles2Nodes_3.resize( coordInfoVecs.triangles2Nodes_3.size() );
	std::cout<<"Host_triangles2Nodes size = "<<hostSetInfoVecs.triangles2Nodes_1.size()<<std::endl;
	
	hostSetInfoVecs.triangles2Edges_1.resize( coordInfoVecs.triangles2Edges_1.size() );
	hostSetInfoVecs.triangles2Edges_2.resize( coordInfoVecs.triangles2Edges_2.size() );
	hostSetInfoVecs.triangles2Edges_3.resize( coordInfoVecs.triangles2Edges_3.size() );
	std::cout<<"Host_triangles2Edges size = "<<hostSetInfoVecs.triangles2Edges_1.size()<<std::endl;

	hostSetInfoVecs.edges2Nodes_1.resize( coordInfoVecs.edges2Nodes_1.size() );
	hostSetInfoVecs.edges2Nodes_2.resize( coordInfoVecs.edges2Nodes_2.size() );
	std::cout<<"Host_edges2Nodes size = "<<hostSetInfoVecs.edges2Nodes_1.size()<<std::endl;
	
	hostSetInfoVecs.edges2Triangles_1.resize( coordInfoVecs.edges2Triangles_1.size() );
	hostSetInfoVecs.edges2Triangles_2.resize( coordInfoVecs.edges2Triangles_2.size() );
	std::cout<<"Host_edges2Triangles size = "<<hostSetInfoVecs.edges2Triangles_1.size()<<std::endl;

	hostSetInfoVecs.nndata1.resize( mem_prealloc*generalParams.maxNodeCount);
	hostSetInfoVecs.nndata2.resize( mem_prealloc*generalParams.maxNodeCount);
	hostSetInfoVecs.nndata3.resize( mem_prealloc*generalParams.maxNodeCount);
	hostSetInfoVecs.nndata4.resize( mem_prealloc*generalParams.maxNodeCount);
	hostSetInfoVecs.nndata5.resize( mem_prealloc*generalParams.maxNodeCount);
	hostSetInfoVecs.nndata6.resize( mem_prealloc*generalParams.maxNodeCount);
	hostSetInfoVecs.nndata7.resize( mem_prealloc*generalParams.maxNodeCount);
	hostSetInfoVecs.nndata8.resize( mem_prealloc*generalParams.maxNodeCount);
	hostSetInfoVecs.nndata9.resize( mem_prealloc*generalParams.maxNodeCount);
	hostSetInfoVecs.nndata10.resize( mem_prealloc*generalParams.maxNodeCount);
	hostSetInfoVecs.nndata11.resize( mem_prealloc*generalParams.maxNodeCount);
	hostSetInfoVecs.nndata12.resize( mem_prealloc*generalParams.maxNodeCount);

	std::cout<<"initial lengths: "<< linearSpringInfoVecs.edge_initial_length.size()<<std::endl;

	std::cout<<"System Ready"<<std::endl;

	//Generate LJ particle list. and set LJ particle midpoint.
	//double maxX_lj = *(thrust::max_element(coordInfoVecs.nodeLocX.begin(),coordInfoVecs.nodeLocX.end()));
	//double minX_lj = *(thrust::min_element(coordInfoVecs.nodeLocX.begin(),coordInfoVecs.nodeLocX.end()));
	//double maxY_lj = *(thrust::max_element(coordInfoVecs.nodeLocY.begin(),coordInfoVecs.nodeLocY.end()));
	//double minY_lj = *(thrust::min_element(coordInfoVecs.nodeLocY.begin(),coordInfoVecs.nodeLocY.end()));
	
	//ljInfoVecs.LJ_PosX = (maxX_lj + minX_lj)/2.0;
	//ljInfoVecs.LJ_PosY = (maxY_lj + minY_lj)/2.0;


	//currently unused
	/*thrust::host_vector<int> tempIds;
	for (int i = 0; i < hostSetInfoVecs.nodeLocX.size(); i++ ) {
		double xLoc = hostSetInfoVecs.nodeLocX[i];
		double yLoc = hostSetInfoVecs.nodeLocY[i];
		double zLoc = hostSetInfoVecs.nodeLocZ[i];
		
		double xDist = ljInfoVecs.LJ_PosX - xLoc;
		double yDist = ljInfoVecs.LJ_PosY - yLoc;
		double zDist = ljInfoVecs.LJ_PosZ - zLoc;

		double dist = std::sqrt(xDist*xDist + yDist*yDist + zDist*zDist);
		//just test all poitns for now. Optimize later.
		if (dist < ljInfoVecs.Rcutoff) {
			tempIds.push_back(i);
		}
	}
	ljInfoVecs.node_id_close.resize( tempIds.size() );
	thrust::copy(tempIds.begin(), tempIds.end(), ljInfoVecs.node_id_close.begin());
	std::cout<<"lj nodes: "<< ljInfoVecs.node_id_close.size() << std::endl;*/






	//last, set memory foor buckets.
	auxVecs.id_bucket.resize(generalParams.maxNodeCount);
	auxVecs.id_value.resize(generalParams.maxNodeCount);
	auxVecs.id_bucket_expanded.resize(27 * (generalParams.maxNodeCount));
	auxVecs.id_value_expanded.resize(27 *( generalParams.maxNodeCount ));
 


};


